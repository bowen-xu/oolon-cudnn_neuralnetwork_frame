#include <hip/hip_runtime.h>
#include <>

#include <hipblas.h>
#include <hipDNN.h>

#include <stdio.h>
#include "neuralnetwork.cuh"

void main()
{
	NeuralNetwork nn;

	nn.AddData(new DataSet());
	nn.AddLayer(new ConvolutionLayer(&nn, nn.Data, 20, 5));
	nn.AddLayer(new MaxPoolLayer(&nn, nn.Layers.back(), 2, 2));
	nn.AddLayer(new ConvolutionLayer(&nn, nn.Layers.back(), 50, 5));
	nn.AddLayer(new MaxPoolLayer(&nn, nn.Layers.back(), 2, 2));
	nn.AddLayer(new FullyConnectedLayer(&nn, nn.Layers.back(), 500));
	nn.AddLayer(new ActivationLayer(&nn, nn.Layers.back()));
	nn.AddLayer(new FullyConnectedLayer(&nn, nn.Layers.back(), 10));
	nn.AddLayer(new OutputLayer(&nn, nn.Layers.back()));

	nn.Create();
	nn.Train(1000);
	nn.Test();
	
	getchar();

}