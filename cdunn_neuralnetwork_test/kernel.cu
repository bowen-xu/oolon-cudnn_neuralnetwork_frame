#include <hip/hip_runtime.h>
#include <>

#include <hipblas.h>
#include <hipDNN.h>

#include <stdio.h>
#include "neuralnetwork.cuh"

void main()
{
	NeuralNetwork nn;

	nn.Create();
	nn.Train(10000);
	nn.Test();
	
	getchar();

}