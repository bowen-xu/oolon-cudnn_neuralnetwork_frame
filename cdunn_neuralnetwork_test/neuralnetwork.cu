#include "hip/hip_runtime.h"
#include "neuralnetwork.cuh"

/**
* Computes ceil(x / y) for integral nonnegative values.
*/
static inline unsigned int RoundUp(unsigned int nominator, unsigned int denominator)
{
	return (nominator + denominator - 1) / denominator;
}

/**
* Fills a floating-point array with ones.
*
* @param vec The array to fill.
* @param size The number of elements in the array.
*/
__global__ void FillOnes(float *vec, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;

	vec[idx] = 1.0f;
}
/**
* Computes the backpropagation results of the Softmax loss for each result in a batch.
* Uses the softmax values obtained from forward propagation to compute the difference.
*
* @param label The training batch label values.
* @param num_labels The number of possible labels.
* @param batch_size The size of the trained batch.
* @param diff The resulting gradient.
*/
__global__ void SoftmaxLossBackprop(const float *label, int num_labels, int batch_size, float *diff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= batch_size)
		return;

	const int label_value = static_cast<int>(label[idx]);

	// For each item in the batch, decrease the result of the label's value by 1
	diff[idx * num_labels + label_value] -= 1.0f;
}

/*
// FullyConnectedLayer
*/
FullyConnectedLayer::FullyConnectedLayer(int input_num, int output_num)
{
	InputNumber = input_num;
	OutputNumber = output_num;
	ParamW.resize(input_num * output_num);
	ParamB.resize(output_num);

	random_device rd;
	mt19937 gen(rd());
	float wfc = sqrt(3.0f / (InputNumber * OutputNumber));
	std::uniform_real_distribution<> dfc(-wfc, wfc);
	for (auto&& iter : ParamW)
		iter = static_cast<float>(dfc(gen));
	for (auto&& iter : ParamB)
		iter = static_cast<float>(dfc(gen));

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);
}

FullyConnectedLayer::~FullyConnectedLayer()
{
	DestroyDescriptor();
	deviceFree();
}


inline void FullyConnectedLayer::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * OutputNumber));	// GPU�и����ݿ��ٿռ�
																							// ����
	checkCudaErrors(hipMalloc(&device_param_w, sizeof(float) * ParamW.size()));			// GPU�и�����w���ٿռ�
	checkCudaErrors(hipMalloc(&device_param_b, sizeof(float) * ParamB.size()));			// GPU�и�����b���ٿռ�
																							// �ݶ�
	checkCudaErrors(hipMalloc(&device_grad_w, sizeof(float) * ParamW.size()));			// GPU�и��ݶ�w���ٿռ�
	checkCudaErrors(hipMalloc(&device_grad_b, sizeof(float) * ParamB.size()));			// GPU�и��ݶ�b���ٿռ�
																						// ���򴫲�����
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * InputNumber));

	// ������ʼ�����ݵ�GPU
	checkCudaErrors(hipMemcpyAsync(device_param_w, &ParamW[0], sizeof(float) * ParamW.size(), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyAsync(device_param_b, &ParamB[0], sizeof(float) * ParamB.size(), hipMemcpyHostToDevice));
}

inline void FullyConnectedLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_param_w));
	checkCudaErrors(hipFree(device_param_b));
	checkCudaErrors(hipFree(device_grad_w));
	checkCudaErrors(hipFree(device_grad_b));
	checkCudaErrors(hipFree(device_diff_data));
	checkCudaErrors(hipFree(device_param_w));
	checkCudaErrors(hipFree(device_param_b));
}

inline void FullyConnectedLayer::CreateDescriptor(int batchsize)
{
	checkCUDNN(hipdnnCreateTensorDescriptor(&TensorDesc));

	//
	checkCUDNN(hipdnnSetTensor4dDescriptor(TensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchsize, OutputNumber, 1, 1));
}

inline void FullyConnectedLayer::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(TensorDesc));
}


/*
// ActivationLayer
*/
ActivationLayer::ActivationLayer(int num, hipdnnActivationMode_t mode, hipdnnNanPropagation_t nanopt, double coef)
{
	Number = num;
	ActivationMode = mode;
	NanOption = nanopt;
	Coef = coef;

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);
}

ActivationLayer::~ActivationLayer()
{
	DestroyDescriptor();
	deviceFree();
}

inline void ActivationLayer::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * Number));
	// ���򴫲�����
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * Number));
}

inline void ActivationLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_diff_data));
}

inline void ActivationLayer::CreateDescriptor(int batchsize)
{
	// ����������
	checkCUDNN(hipdnnCreateActivationDescriptor(&ActivationDesc));

	// ����������
	checkCUDNN(hipdnnSetActivationDescriptor(ActivationDesc, ActivationMode, NanOption, Coef));
}

inline void ActivationLayer::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyActivationDescriptor(ActivationDesc));
}



/*
// ConvolutionLayer
*/
ConvolutionLayer::ConvolutionLayer(hipdnnHandle_t cudnnhandle, hipdnnTensorDescriptor_t lastTensorDesc, int in_channels, int out_channels, int kernel_size, int in_width, int in_height, int padding, int stride)
{
	InputChannels = in_channels;
	OutputChannels = out_channels;
	KernelSize = kernel_size;
	InputWidth = in_width;
	InputHeight = in_height;
	Padding = padding;
	Stride = stride;

	OutputHeight = (InputHeight + 2 * Padding - KernelSize) / Stride + 1;
	OutputWidth = (InputWidth + 2 * Padding - KernelSize) / Stride + 1;
	ParamW.resize(in_channels * kernel_size * kernel_size * out_channels);
	ParamB.resize(out_channels);

	LastTensorDesc = lastTensorDesc;
	cudnnHandle = cudnnhandle;

	random_device rd;
	mt19937 gen(rd());
	float wconv = sqrt(3.0f / (KernelSize * KernelSize * InputChannels));
	std::uniform_real_distribution<> dconv(-wconv, wconv);
	for (auto&& iter : ParamW)
		iter = static_cast<float>(dconv(gen));
	for (auto&& iter : ParamB)
		iter = static_cast<float>(dconv(gen));

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);

}

ConvolutionLayer::~ConvolutionLayer()
{
	DestroyDescriptor();
	deviceFree();
}


inline void ConvolutionLayer::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * OutputChannels * OutputHeight * OutputWidth));

	// ����
	checkCudaErrors(hipMalloc(&device_param_w, sizeof(float) * ParamW.size()));
	checkCudaErrors(hipMalloc(&device_param_b, sizeof(float) * ParamB.size()));
	// �ݶ�
	checkCudaErrors(hipMalloc(&device_grad_w, sizeof(float) * ParamW.size()));
	checkCudaErrors(hipMalloc(&device_grad_b, sizeof(float) * ParamB.size()));
	// ���򴫲�����
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * OutputChannels * InputWidth * InputHeight));

	// ������ʼ�����ݵ�GPU
	checkCudaErrors(hipMemcpyAsync(device_param_w, &ParamW[0], sizeof(float) * ParamW.size(), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyAsync(device_param_b, &ParamB[0], sizeof(float) * ParamB.size(), hipMemcpyHostToDevice));
}

inline void ConvolutionLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_param_w));
	checkCudaErrors(hipFree(device_param_b));
	checkCudaErrors(hipFree(device_grad_w));
	checkCudaErrors(hipFree(device_grad_b));
	checkCudaErrors(hipFree(device_diff_data));
	checkCudaErrors(hipFree(device_param_w));
	checkCudaErrors(hipFree(device_param_b));

}

inline void ConvolutionLayer::CreateDescriptor(int batchsize)
{
	size_t tempsize;
	// ��������
	checkCUDNN(hipdnnCreateTensorDescriptor(&TensorDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&BiasTensorDesc));
	checkCUDNN(hipdnnCreateFilterDescriptor(&FilterDesc));
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&ConvDesc));

	// ��������
	checkCUDNN(hipdnnSetTensor4dDescriptor(BiasTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, OutputChannels, 1, 1));
	checkCUDNN(hipdnnSetFilter4dDescriptor(FilterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, OutputChannels, InputChannels, KernelSize, KernelSize));
	checkCUDNN(hipdnnSetConvolution2dDescriptor(ConvDesc, Padding, Padding, Stride, Stride, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
	checkCUDNN(hipdnnSetTensor4dDescriptor(TensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchsize, OutputChannels, OutputHeight, OutputWidth));

	// ǰ�򴫲��㷨����
	checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnnHandle, LastTensorDesc, FilterDesc, ConvDesc, TensorDesc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &FwdAlgDesc));
	checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle, LastTensorDesc, FilterDesc, ConvDesc, TensorDesc, FwdAlgDesc, &tempsize));
	WorkspaceSize = max(WorkspaceSize, tempsize);

	// ���򴫲��㷨����
	checkCUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(cudnnHandle, LastTensorDesc, TensorDesc, ConvDesc, FilterDesc, HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &BwdAlgDesc));
	checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cudnnHandle, LastTensorDesc, TensorDesc, ConvDesc, FilterDesc, BwdAlgDesc, &tempsize));
	WorkspaceSize = max(WorkspaceSize, tempsize);
	checkCUDNN(hipdnnGetConvolutionBackwardDataAlgorithm(cudnnHandle, FilterDesc, TensorDesc, ConvDesc, LastTensorDesc, HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &BwdDataAlgDesc));
	checkCUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnnHandle, FilterDesc, TensorDesc, ConvDesc, LastTensorDesc, BwdDataAlgDesc, &tempsize));
	WorkspaceSize = max(WorkspaceSize, tempsize);
}

inline void ConvolutionLayer::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(TensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(BiasTensorDesc));
	checkCUDNN(hipdnnDestroyFilterDescriptor(FilterDesc));
	checkCUDNN(hipdnnDestroyConvolutionDescriptor(ConvDesc));
}

/*
// MaxPoolLayer
*/
MaxPoolLayer::MaxPoolLayer(int size, int stride, ConvolutionLayer &lastConv)
{
	Size = size;
	Stride = stride;
	OutputChannels = lastConv.OutputChannels;
	OutputWidth = (lastConv.OutputWidth / stride);
	OutputHeight = (lastConv.OutputHeight / stride);

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);
}

MaxPoolLayer::~MaxPoolLayer()
{
	DestroyDescriptor();
	deviceFree();
}


inline void MaxPoolLayer::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * OutputChannels * OutputHeight * OutputWidth));
	// ���򴫲�����
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * OutputChannels * OutputHeight * OutputWidth));
}

inline void MaxPoolLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_diff_data));
}

inline void MaxPoolLayer::CreateDescriptor(int batchsize)
{
	// ����������
	checkCUDNN(hipdnnCreateTensorDescriptor(&TensorDesc));
	checkCUDNN(hipdnnCreatePoolingDescriptor(&PoolDesc));

	// ����������
	checkCUDNN(hipdnnSetPooling2dDescriptor(PoolDesc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN, Size, Size, 0, 0, Stride, Stride));
	checkCUDNN(hipdnnSetTensor4dDescriptor(TensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchsize, OutputChannels, OutputHeight, OutputWidth));
}

inline void MaxPoolLayer::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(TensorDesc));
	checkCUDNN(hipdnnDestroyPoolingDescriptor(PoolDesc));
}


/*
// DataSet
*/
DataSet::DataSet()
{
	printf("Reading input data\n");

	size_t train_size = ReadUByteDataset(TrainingSetName.c_str(), TrainingLabelsName.c_str(), nullptr, nullptr, Width, Height);
	size_t test_size = ReadUByteDataset(TestSetName.c_str(), TestLabelsName.c_str(), nullptr, nullptr, Width, Height);
	TrainSize = train_size;
	TestSize = test_size;
	if (train_size == 0)
		exit(1);

	TrainSet.resize(train_size * Width * Height * Channels);
	TrainLabels.resize(train_size);
	TestSet.resize(test_size * Width * Height * Channels);
	TestLabels.resize(test_size);

	if (ReadUByteDataset(TrainingSetName.c_str(), TrainingLabelsName.c_str(), &TrainSet[0], &TrainLabels[0], Width, Height) != train_size)
		exit(2);
	if (ReadUByteDataset(TestSetName.c_str(), TestLabelsName.c_str(), &TestSet[0], &TestLabels[0], Width, Height) != test_size)
		exit(3);

	printf("Done. Training dataset size: %d, Test dataset size: %d\n", (int)train_size, (int)test_size);
	
	// Normalize training set to be in [0,1]
	printf("Normalizing training data...\n");
	TrainSet_float.resize(TrainSet.size());
	TrainLabels_float.resize(TrainLabels.size());
	for (size_t i = 0; i < train_size * Channels * Width * Height; ++i)
		TrainSet_float[i] = (float)TrainSet[i] / 255.0f;

	for (size_t i = 0; i < train_size; ++i)
		TrainLabels_float[i] = (float)TrainLabels[i];

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);
}

DataSet::~DataSet()
{
	DestroyDescriptor();
	deviceFree();
}

inline void DataSet::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * Channels * Height * Width));
	checkCudaErrors(hipMalloc(&device_labels, sizeof(float) * batchsize));
}

inline void DataSet::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_labels));
}

inline void DataSet::CreateDescriptor(int batchsize)
{
	// ��������
	checkCUDNN(hipdnnCreateTensorDescriptor(&TensorDesc)); //

														  // ��������
	checkCUDNN(hipdnnSetTensor4dDescriptor(TensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchsize, Channels, Height, Width));
}

inline void DataSet::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(TensorDesc));
}

/*
// OutputLayer
*/
OutputLayer::OutputLayer(int num)
{
	Number = num;

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);
}

OutputLayer::~OutputLayer()
{
	DestroyDescriptor();
	deviceFree();
}

inline void OutputLayer::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * Number));
	// ���򴫲�����
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * Number));
	checkCudaErrors(hipMalloc(&device_loss_data, sizeof(float) * batchsize * Number));
}

inline void OutputLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_diff_data));
	checkCudaErrors(hipFree(device_loss_data));
}

inline void OutputLayer::CreateDescriptor(int batchsize)
{

}

inline void OutputLayer::DestroyDescriptor()
{

}

/*
// NeuralNetwork
*/
NeuralNetwork::NeuralNetwork()
{
	int gpu_num;
	checkCudaErrors(hipGetDeviceCount(&gpu_num));
	if (GPUid < 0 || GPUid >= gpu_num)
	{
		printf("ERROR: Invalid GPU ID %d (There are %d GPUs on this machine)\n", GPUid, gpu_num);
		exit(4);
	}

	checkCudaErrors(hipblasCreate(&cublasHandle));
	checkCUDNN(hipdnnCreate(&cudnnHandle));
}

void NeuralNetwork::Create()
{
	Image = new DataSet();
	Conv1 = new ConvolutionLayer(cudnnHandle, Image->TensorDesc, Image->Channels, 20, 5, Image->Height, Image->Width);
	Pool1 = new MaxPoolLayer(2, 2, *Conv1);
	Conv2 = new ConvolutionLayer(cudnnHandle, Pool1->TensorDesc, Conv1->OutputChannels, 50, 5, Conv1->OutputWidth / Pool1->Stride, Conv1->OutputHeight / Pool1->Stride);
	Pool2 = new MaxPoolLayer(2, 2, *Conv2);
	FC1 = new FullyConnectedLayer((Conv2->OutputChannels * Conv2->OutputWidth * Conv2->OutputHeight) / (Pool2->Stride * Pool2->Stride), 500);
	ACTN1 = new ActivationLayer(FC1->OutputNumber);
	FC2 = new FullyConnectedLayer(FC1->OutputNumber, 10);
	RSLT = new OutputLayer(FC2->OutputNumber);
	
	checkCudaErrors(hipMalloc(&device_ones, sizeof(float)* BATCH_SIZE));
	FillOnes <<<RoundUp(BATCH_SIZE, BW), BW>>> (device_ones, BATCH_SIZE);
	WorkspaceSize = max(Conv1->WorkspaceSize, Conv2->WorkspaceSize);
	if (WorkspaceSize > 0)
		checkCudaErrors(hipMalloc(&device_workspace, WorkspaceSize));

}

void NeuralNetwork::Destroy()
{
	delete Image;
	delete Conv1;
	delete Pool1;
	delete Conv2;
	delete Pool2;
	delete FC1;
	delete ACTN1;
	delete FC2;
	delete RSLT;

	checkCudaErrors(hipFree(device_ones));
	if (device_workspace != nullptr)
		checkCudaErrors(hipFree(device_workspace));
	checkCudaErrors(hipDeviceReset());
}

void NeuralNetwork::Train(int iterations)
{
	printf("Training...\n");
	checkCudaErrors(hipDeviceSynchronize());
	auto t1 = std::chrono::high_resolution_clock::now();
	size_t train_size = Image->TrainSize;
	for (int iter = 0; iter < iterations; ++iter)
	{
		// Train
		int imageid = iter % (train_size / BATCH_SIZE);

		// Prepare current batch on device
		checkCudaErrors(hipMemcpyAsync(Image->device_data, &((Image->TrainSet_float)[imageid * BATCH_SIZE * Image->Width * Image->Height * Image->Channels]),
			sizeof(float) * BATCH_SIZE * Image->Width *Image->Height * Image->Channels, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpyAsync(Image->device_labels, &((Image->TrainLabels_float)[imageid * BATCH_SIZE]),
			sizeof(float) * BATCH_SIZE, hipMemcpyHostToDevice));

		// Forward propagation
		ForwardPropagate();

		//std::vector<float> class_vec(10);
		//// Copy back result
		//checkCudaErrors(hipMemcpy(&class_vec[0], RSLT->device_data, sizeof(float) * 10, hipMemcpyDeviceToHost));


		// Backward propagation
		BackPropagate();

		// Compute learning rate
		float learningRate = static_cast<float>(LEARNING_RATE * pow((1.0 + LR_GAMMA * iter), (-LR_POWER)));

		// Update weights
		UpdateWeights(learningRate);
	}
	checkCudaErrors(hipDeviceSynchronize());
	auto t2 = std::chrono::high_resolution_clock::now();

	printf("Iteration time: %f ms\n", std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count() / 1000.0f / iterations);

}


void NeuralNetwork::Test()
{
	float classification_error = 1.0f;

	int classifications = (int)(Image->TestSize);

	// Test the resulting neural network's classification

	// Initialize a TrainingContext structure for testing (different batch size)
	NeuralNetwork test_nn;

	// Ensure correct workspaceSize is allocated for testing
	if (WorkspaceSize < test_nn.WorkspaceSize)
	{
		checkCudaErrors(hipFree(device_workspace));
		checkCudaErrors(hipMalloc(&device_workspace, test_nn.WorkspaceSize));
	}

	int num_errors = 0;
	for (int i = 0; i < classifications; ++i)
	{
		std::vector<float> data(Image->Width * Image->Height);
		// Normalize image to be in [0,1]
		for (int j = 0; j < Image->Width * Image->Height; ++j)
			data[j] = (float)Image->TestSet[i * Image->Width * Image->Height * Image->Channels + j] / 255.0f;

		checkCudaErrors(hipMemcpyAsync(Image->device_data, &data[0], sizeof(float) * Image->Width * Image->Height, hipMemcpyHostToDevice));

		// Forward propagate test image
		ForwardPropagate();

		// Perform classification
		std::vector<float> class_vec(10);

		// Copy back result
		checkCudaErrors(hipMemcpy(&class_vec[0], RSLT->device_data, sizeof(float) * 10, hipMemcpyDeviceToHost));

		// Determine classification according to maximal response
		int chosen = 0;
		for (int id = 1; id < 10; ++id)
		{
			if (class_vec[chosen] < class_vec[id]) chosen = id;
		}

		if (chosen != Image->TestLabels[i])
			++num_errors;
	}
	classification_error = (float)num_errors / (float)classifications;

	printf("Classification result: %.2f%% error (used %d images)\n", classification_error * 100.0f, (int)classifications);
	
}


void NeuralNetwork::ForwardPropagate()
{
	float alpha = 1.0f, beta = 0.0f;
	checkCudaErrors(hipSetDevice(GPUid));

	// Conv1 layer
	checkCUDNN(hipdnnConvolutionForward(cudnnHandle, &alpha, Image->TensorDesc,
		Image->device_data, Conv1->FilterDesc, Conv1->device_param_w, Conv1->ConvDesc,
		Conv1->FwdAlgDesc, device_workspace, WorkspaceSize, &beta,
		Conv1->TensorDesc, Conv1->device_data));

	checkCUDNN(hipdnnAddTensor(cudnnHandle, &alpha, Conv1->BiasTensorDesc,
		Conv1->device_param_b, &alpha, Conv1->TensorDesc, Conv1->device_data));

	// Pool1 layer
	checkCUDNN(hipdnnPoolingForward(cudnnHandle, Pool1->PoolDesc, &alpha, Conv1->TensorDesc,
		Conv1->device_data, &beta, Pool1->TensorDesc, Pool1->device_data));

	// Conv2 layer
	checkCUDNN(hipdnnConvolutionForward(cudnnHandle, &alpha, Pool1->TensorDesc,
		Pool1->device_data, Conv2->FilterDesc, Conv2->device_param_w, Conv2->ConvDesc,
		Conv2->FwdAlgDesc, device_workspace, WorkspaceSize, &beta,
		Conv2->TensorDesc, Conv2->device_data));
	checkCUDNN(hipdnnAddTensor(cudnnHandle, &alpha, Conv2->BiasTensorDesc,
		Conv2->device_param_b, &alpha, Conv2->TensorDesc, Conv2->device_data));

	// Pool2 layer
	checkCUDNN(hipdnnPoolingForward(cudnnHandle, Pool2->PoolDesc, &alpha, Conv2->TensorDesc,
		Conv2->device_data, &beta, Pool2->TensorDesc, Pool2->device_data));

	// FC1 layer
	// Forward propagate neurons using weights (fc1 = pfc1'*pool2)
	checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
		FC1->OutputNumber, BATCH_SIZE, FC1->InputNumber,
		&alpha,
		FC1->device_param_w, FC1->InputNumber,
		Pool2->device_data, FC1->InputNumber,
		&beta,
		FC1->device_data, FC1->OutputNumber));
	// Add bias using GEMM's "beta" (fc1 += pfc1bias*1_vec')
	checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		FC1->OutputNumber, BATCH_SIZE, 1,
		&alpha,
		FC1->device_param_b, FC1->OutputNumber,
		device_ones, 1,
		&alpha,
		FC1->device_data, FC1->OutputNumber));

	// ReLU activation
	checkCUDNN(hipdnnActivationForward(cudnnHandle, ACTN1->ActivationDesc, &alpha,
		FC1->TensorDesc, FC1->device_data, &beta, FC1->TensorDesc, ACTN1->device_data));

	// FC2 layer
	// Forward propagate neurons using weights (fc2 = pfc2'*fc1relu)
	checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
		FC2->OutputNumber, BATCH_SIZE, FC2->InputNumber,
		&alpha,
		FC2->device_param_w, FC2->InputNumber,
		ACTN1->device_data, FC2->InputNumber,
		&beta,
		FC2->device_data, FC2->OutputNumber));
	// Add bias using GEMM's "beta" (fc2 += pfc2bias*1_vec')
	checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		FC2->OutputNumber, BATCH_SIZE, 1,
		&alpha,
		FC2->device_param_b, FC2->OutputNumber,
		device_ones, 1,
		&alpha,
		FC2->device_data, FC2->OutputNumber));

	// Softmax loss
	checkCUDNN(hipdnnSoftmaxForward(cudnnHandle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
		&alpha, FC2->TensorDesc, FC2->device_data, &beta, FC2->TensorDesc, RSLT->device_data));
}

void NeuralNetwork::BackPropagate()
{
	float alpha = 1.0f, beta = 0.0f;

	float scalVal = 1.0f / static_cast<float>(BATCH_SIZE);

	checkCudaErrors(hipSetDevice(GPUid));

	// Initialization (using the training error function)
	checkCudaErrors(hipMemcpyAsync(RSLT->device_diff_data, RSLT->device_data, sizeof(float) * BATCH_SIZE * FC2->OutputNumber, hipMemcpyDeviceToDevice));

	// Softmax layer
	SoftmaxLossBackprop <<<RoundUp(BATCH_SIZE, BW), BW >>> (Image->device_labels, FC2->OutputNumber, BATCH_SIZE, RSLT->device_diff_data);

	// Accounting for batch size in SGD
	checkCudaErrors(hipblasSscal(cublasHandle, FC2->OutputNumber * BATCH_SIZE, &scalVal, RSLT->device_diff_data, 1));

	// FC2 layer
	// Compute derivative with respect to weights: gfc2 = (fc1relu * dfc2smax')
	checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, FC2->InputNumber, FC2->OutputNumber, BATCH_SIZE,
		&alpha, ACTN1->device_data, FC2->InputNumber, RSLT->device_diff_data, FC2->OutputNumber, &beta, FC2->device_grad_w, FC2->InputNumber));
	// Compute derivative with respect to bias: gfc2bias = dfc2smax * 1_vec
	checkCudaErrors(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, FC2->OutputNumber, BATCH_SIZE,
		&alpha, RSLT->device_diff_data, FC2->OutputNumber, device_ones, 1, &beta, FC2->device_grad_b, 1));
	// Compute derivative with respect to data (for previous layer): pfc2*dfc2smax (500x10*10xN)
	checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, FC2->InputNumber, BATCH_SIZE, FC2->OutputNumber,
		&alpha, FC2->device_param_w, FC2->InputNumber, RSLT->device_diff_data, FC2->OutputNumber, &beta, FC2->device_diff_data, FC2->InputNumber));

	// ReLU activation
	checkCUDNN(hipdnnActivationBackward(cudnnHandle, ACTN1->ActivationDesc, &alpha,
		FC1->TensorDesc, ACTN1->device_data, FC1->TensorDesc, FC2->device_diff_data,
		FC1->TensorDesc, FC1->device_data, &beta, FC1->TensorDesc, ACTN1->device_diff_data));

	// FC1 layer
	// Compute derivative with respect to weights: gfc1 = (pool2 * dfc1relu')
	checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, FC1->InputNumber, FC1->OutputNumber, BATCH_SIZE,
		&alpha, Pool2->device_data, FC1->InputNumber, ACTN1->device_diff_data, FC1->OutputNumber, &beta, FC1->device_grad_w, FC1->InputNumber));
	// Compute derivative with respect to bias: gfc1bias = dfc1relu * 1_vec
	checkCudaErrors(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, FC1->OutputNumber, BATCH_SIZE,
		&alpha, ACTN1->device_diff_data, FC1->OutputNumber, device_ones, 1, &beta, FC1->device_grad_b, 1));
	// Compute derivative with respect to data (for previous layer): pfc1*dfc1relu (800x500*500xN)
	checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, FC1->InputNumber, BATCH_SIZE, FC1->OutputNumber,
		&alpha, FC1->device_param_w, FC1->InputNumber, ACTN1->device_diff_data, FC1->OutputNumber, &beta, FC1->device_diff_data, FC1->InputNumber));

	// Pool2 layer
	checkCUDNN(hipdnnPoolingBackward(cudnnHandle, Pool2->PoolDesc, &alpha,
		Pool2->TensorDesc, Pool2->device_data, Pool2->TensorDesc, FC1->device_diff_data,
		Conv2->TensorDesc, Conv2->device_data, &beta, Conv2->TensorDesc, Pool2->device_diff_data));

	// Conv2 layer
	checkCUDNN(hipdnnConvolutionBackwardBias(cudnnHandle, &alpha, Conv2->TensorDesc,
		Pool2->device_diff_data, &beta, Conv2->BiasTensorDesc, Conv2->device_grad_b));


	checkCUDNN(hipdnnConvolutionBackwardFilter(cudnnHandle, &alpha, Pool1->TensorDesc,
		Pool1->device_data, Conv2->TensorDesc, Pool2->device_diff_data, Conv2->ConvDesc,
		Conv2->BwdAlgDesc, device_workspace, WorkspaceSize,
		&beta, Conv2->FilterDesc, Conv2->device_grad_w));

	checkCUDNN(hipdnnConvolutionBackwardData(cudnnHandle, &alpha, Conv2->FilterDesc,
		Conv2->device_param_w, Conv2->TensorDesc, Pool2->device_diff_data, Conv2->ConvDesc,
		Conv2->BwdDataAlgDesc, device_workspace, WorkspaceSize,
		&beta, Pool1->TensorDesc, Conv2->device_diff_data));

	// Pool1 layer
	checkCUDNN(hipdnnPoolingBackward(cudnnHandle, Pool1->PoolDesc, &alpha,
		Pool1->TensorDesc, Pool1->device_data, Pool1->TensorDesc, Conv2->device_diff_data,
		Conv1->TensorDesc, Conv1->device_data, &beta, Conv1->TensorDesc, Pool1->device_diff_data));

	// Conv1 layer
	checkCUDNN(hipdnnConvolutionBackwardBias(cudnnHandle, &alpha, Conv1->TensorDesc,
		Pool1->device_diff_data, &beta, Conv1->BiasTensorDesc, Conv1->device_grad_b));

	checkCUDNN(hipdnnConvolutionBackwardFilter(cudnnHandle, &alpha, Image->TensorDesc,
		Image->device_data, Conv1->TensorDesc, Pool1->device_diff_data, Conv1->ConvDesc,
		Conv1->BwdAlgDesc, device_workspace, WorkspaceSize,
		&beta, Conv1->FilterDesc, Conv1->device_grad_w));

	// No need for convBackwardData because there are no more layers below
}

void NeuralNetwork::UpdateWeights(float learning_rate)
{
	float alpha = -learning_rate;

	checkCudaErrors(hipSetDevice(GPUid));

	// Conv1
	checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(Conv1->ParamW.size()),
		&alpha, Conv1->device_grad_w, 1, Conv1->device_param_w, 1));
	checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(Conv1->ParamB.size()),
		&alpha, Conv1->device_grad_b, 1, Conv1->device_param_b, 1));

	// Conv2
	checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(Conv2->ParamW.size()),
		&alpha, Conv2->device_grad_w, 1, Conv2->device_param_w, 1));
	checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(Conv2->ParamB.size()),
		&alpha, Conv2->device_grad_b, 1, Conv2->device_param_b, 1));

	// Fully connected 1
	checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(FC1->ParamW.size()),
		&alpha, FC1->device_grad_w, 1, FC1->device_param_w, 1));
	checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(FC1->ParamB.size()),
		&alpha, FC1->device_grad_b, 1, FC1->device_param_b, 1));

	// Fully connected 2
	checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(FC2->ParamW.size()),
		&alpha, FC2->device_grad_w, 1, FC2->device_param_w, 1));
	checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(FC2->ParamB.size()),
		&alpha, FC2->device_grad_b, 1, FC2->device_param_b, 1));
}





