#include "hip/hip_runtime.h"
#include "neuralnetwork.cuh"

/**
* Computes ceil(x / y) for integral nonnegative values.
*/
static inline unsigned int RoundUp(unsigned int nominator, unsigned int denominator)
{
	return (nominator + denominator - 1) / denominator;
}

/**
* Fills a floating-point array with ones.
*
* @param vec The array to fill.
* @param size The number of elements in the array.
*/
__global__ void FillOnes(float *vec, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;

	vec[idx] = 1.0f;
}
/**
* Computes the backpropagation results of the Softmax loss for each result in a batch.
* Uses the softmax values obtained from forward propagation to compute the difference.
*
* @param label The training batch label values.
* @param num_labels The number of possible labels.
* @param batch_size The size of the trained batch.
* @param diff The resulting gradient.
*/
__global__ void SoftmaxLossBackprop(const float *label, const float *data, float *diff, int num_labels, int batch_size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= batch_size)
		return;

	const int label_value = static_cast<int>(label[idx]);

	// For each item in the batch, decrease the result of the label's value by 1
	diff[idx * num_labels + label_value] = data[idx * num_labels + label_value] - 1.0f;
}



/*
// FullyConnectedLayer
*/
FullyConnectedLayer::FullyConnectedLayer(NeuralNetwork *neuralnetwork, Layer *lastlayer, int num)
{
	InputNumber = lastlayer->OutputNumber;
	OutputNumber = num;
	InputChannels = OutputChannels = 1;
	Padding = 0;
	Stride = 1;
	KernelSize = 1;

	ParamW.resize(InputNumber * OutputNumber);
	ParamB.resize(OutputNumber);

	neuralNetwork = neuralnetwork;
	LastLayer = lastlayer;
	lastlayer->NextLayer = this;

	//random_device rd;
	//mt19937 gen(rd());
	//float wfc = sqrt(3.0f / (InputNumber * OutputNumber));
	//std::uniform_real_distribution<> dfc(-wfc, wfc);
	//for (auto&& iter : ParamW)
	//	iter = static_cast<float>(dfc(gen));
	//for (auto&& iter : ParamB)
	//	iter = static_cast<float>(dfc(gen));

	static int index = 0;
	index++;
	char str[256];
	sprintf(str, "FC%d", index);

	if (isSave)
	{
		random_device rd;
		mt19937 gen(rd());
		float wfc = sqrt(3.0f / (InputNumber * OutputNumber));
		std::uniform_real_distribution<> dfc(-wfc, wfc);
		for (auto&& iter : ParamW)
			iter = static_cast<float>(dfc(gen));
		for (auto&& iter : ParamB)
			iter = static_cast<float>(dfc(gen));

		ToFile(str);
	}
	else
	{
		FromFile(str);
	}

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);
}


FullyConnectedLayer::~FullyConnectedLayer()
{
	DestroyDescriptor();
	deviceFree();
}

inline void FullyConnectedLayer::ForwardPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;
	// Forward propagate neurons using weights (fc1 = pfc1'*pool2)
	checkCudaErrors(hipblasSgemm(neuralNetwork->cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
		OutputNumber, BATCH_SIZE, InputNumber, &alpha, device_param_w, InputNumber,
		LastLayer->device_data, InputNumber, &beta, device_data, OutputNumber));
	// Add bias using GEMM's "beta" (fc1 += pfc1bias*1_vec')
	checkCudaErrors(hipblasSgemm(neuralNetwork->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		OutputNumber, BATCH_SIZE, 1, &alpha, device_param_b, OutputNumber, neuralNetwork->device_ones, 1, &alpha,
		device_data, OutputNumber));
}

inline void FullyConnectedLayer::BackPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;
	
	checkCudaErrors(hipblasSgemm(neuralNetwork->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, InputNumber, OutputNumber, BATCH_SIZE,
		&alpha, LastLayer->device_data, InputNumber, NextLayer->device_diff_data, OutputNumber, &beta, device_grad_w, InputNumber));
	
	checkCudaErrors(hipblasSgemv(neuralNetwork->cublasHandle, HIPBLAS_OP_N, OutputNumber, BATCH_SIZE,
		&alpha, NextLayer->device_diff_data, OutputNumber, neuralNetwork->device_ones, 1, &beta, device_grad_b, 1));
	// Compute derivative with respect to data (for previous layer): pfc2*dfc2smax (500x10*10xN)
	if (!isFirstLayer)
	{
		checkCudaErrors(hipblasSgemm(neuralNetwork->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, InputNumber, BATCH_SIZE, OutputNumber,
			&alpha, device_param_w, InputNumber, NextLayer->device_diff_data, OutputNumber, &beta, device_diff_data, InputNumber));// ���һ��device_diff_data ��Ϊdevice_diff_data
	}
}

inline void FullyConnectedLayer::UpdateWeights(float learning_rate)
{
	float alpha = -learning_rate;

	checkCudaErrors(hipblasSaxpy(neuralNetwork->cublasHandle, static_cast<int>(ParamW.size()),
		&alpha, device_grad_w, 1, device_param_w, 1));
	checkCudaErrors(hipblasSaxpy(neuralNetwork->cublasHandle, static_cast<int>(ParamB.size()),
		&alpha, device_grad_b, 1, device_param_b, 1));
}

inline void FullyConnectedLayer::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * OutputNumber));	// GPU�и����ݿ��ٿռ�
																							// ����
	checkCudaErrors(hipMalloc(&device_param_w, sizeof(float) * ParamW.size()));			// GPU�и�����w���ٿռ�
	checkCudaErrors(hipMalloc(&device_param_b, sizeof(float) * ParamB.size()));			// GPU�и�����b���ٿռ�
																							// �ݶ�
	checkCudaErrors(hipMalloc(&device_grad_w, sizeof(float) * ParamW.size()));				// GPU�и��ݶ�w���ٿռ�
	checkCudaErrors(hipMalloc(&device_grad_b, sizeof(float) * ParamB.size()));				// GPU�и��ݶ�b���ٿռ�
																							// ���򴫲�����
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * InputNumber));

	// ������ʼ�����ݵ�GPU
	checkCudaErrors(hipMemcpyAsync(device_param_w, &ParamW[0], sizeof(float) * ParamW.size(), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyAsync(device_param_b, &ParamB[0], sizeof(float) * ParamB.size(), hipMemcpyHostToDevice));
}

inline void FullyConnectedLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_param_w));
	checkCudaErrors(hipFree(device_param_b));
	checkCudaErrors(hipFree(device_grad_w));
	checkCudaErrors(hipFree(device_grad_b));
	checkCudaErrors(hipFree(device_diff_data));
	checkCudaErrors(hipFree(device_param_w));
	checkCudaErrors(hipFree(device_param_b));
}

inline void FullyConnectedLayer::CreateDescriptor(int batchsize)
{
	checkCUDNN(hipdnnCreateTensorDescriptor(&TensorDesc));

	//
	checkCUDNN(hipdnnSetTensor4dDescriptor(TensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchsize, OutputNumber, 1, 1));
}

inline void FullyConnectedLayer::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(TensorDesc));
}


/*
// ActivationLayer
*/
ActivationLayer::ActivationLayer(NeuralNetwork *neuralnetwork, Layer *lastlayer, hipdnnActivationMode_t mode, hipdnnNanPropagation_t nanopt, double coef)
{
	InputNumber = OutputNumber = lastlayer->OutputNumber;
	InputHeight = OutputHeight = lastlayer->OutputHeight;
	InputWidth = OutputWidth = lastlayer->OutputWidth;
	InputChannels = OutputChannels = lastlayer->OutputChannels;
	Padding = 0;
	KernelSize = 1;
	Stride = 1;

	ActivationMode = mode;
	NanOption = nanopt;
	Coef = coef;

	neuralNetwork = neuralnetwork;
	LastLayer = lastlayer;
	lastlayer->NextLayer = this;

	TensorDesc = LastLayer->TensorDesc;

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);
}

ActivationLayer::~ActivationLayer()
{
	DestroyDescriptor();
	deviceFree();
}

inline void ActivationLayer::ForwardPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;
	checkCUDNN(hipdnnActivationForward(neuralNetwork->cudnnHandle, ActivationDesc, &alpha,
		LastLayer->TensorDesc, LastLayer->device_data, &beta, LastLayer->TensorDesc, device_data));
}

inline void ActivationLayer::BackPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;
	if (!isFirstLayer)
	{
		checkCUDNN(hipdnnActivationBackward(neuralNetwork->cudnnHandle, ActivationDesc, &alpha,
			LastLayer->TensorDesc, device_data, LastLayer->TensorDesc, NextLayer->device_diff_data,
			LastLayer->TensorDesc, LastLayer->device_data, &beta, LastLayer->TensorDesc, device_diff_data));
	}
}

inline void ActivationLayer::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * OutputNumber));
	// ���򴫲�����
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * InputNumber));
}

inline void ActivationLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_diff_data));
}

inline void ActivationLayer::CreateDescriptor(int batchsize)
{
	// ����������
	checkCUDNN(hipdnnCreateActivationDescriptor(&ActivationDesc));

	// ����������
	checkCUDNN(hipdnnSetActivationDescriptor(ActivationDesc, ActivationMode, NanOption, Coef));
}

inline void ActivationLayer::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyActivationDescriptor(ActivationDesc));
}



/*
// ConvolutionLayer
*/
ConvolutionLayer::ConvolutionLayer(NeuralNetwork *neuralnetwork, Layer *lastlayer, int output_channels, int kernel_size, int padding, int stride)
{
	InputWidth = lastlayer->OutputWidth;
	InputHeight = lastlayer->OutputHeight;
	OutputHeight = (InputHeight + 2 * padding - kernel_size) / stride + 1;
	OutputWidth = (InputWidth + 2 * padding - kernel_size) / stride + 1;
	InputChannels = lastlayer->OutputChannels;
	OutputChannels = output_channels;
	InputNumber = InputHeight * InputWidth * InputChannels;
	OutputNumber = OutputHeight * OutputWidth * OutputChannels;
	KernelSize = kernel_size;
	Padding = padding;
	Stride = stride;
	
	ParamW.resize(InputChannels * KernelSize * KernelSize * OutputChannels);
	ParamB.resize(OutputChannels);

	neuralNetwork = neuralnetwork;
	LastLayer = lastlayer;
	lastlayer->NextLayer = this;

	static int index = 0;
	index++;
	char str[256];
	sprintf(str, "Conv%d", index);

	if (isSave)
	{
		random_device rd;
		mt19937 gen(rd());
		float wconv = sqrt(3.0f / (KernelSize * KernelSize * InputChannels));
		std::uniform_real_distribution<> dconv(-wconv, wconv);
		for (auto&& iter : ParamW)
			iter = static_cast<float>(dconv(gen));
		for (auto&& iter : ParamB)
			iter = static_cast<float>(dconv(gen));

		ToFile(str);
	}
	else
	{

		FromFile(str);
	}

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);

}

ConvolutionLayer::~ConvolutionLayer()
{
	DestroyDescriptor();
	deviceFree();
}


inline void ConvolutionLayer::ForwardPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;
	checkCUDNN(hipdnnConvolutionForward(
		neuralNetwork->cudnnHandle, &alpha, LastLayer->TensorDesc,
		LastLayer->device_data, FilterDesc, device_param_w, ConvDesc,
		FwdAlgDesc, neuralNetwork->device_workspace, neuralNetwork->WorkspaceSize, &beta,
		TensorDesc, device_data));

	checkCUDNN(hipdnnAddTensor(neuralNetwork->cudnnHandle, &alpha, BiasTensorDesc,
		device_param_b, &alpha, TensorDesc, device_data));
}

inline void ConvolutionLayer::BackPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;

	checkCUDNN(hipdnnConvolutionBackwardBias(neuralNetwork->cudnnHandle, &alpha, TensorDesc,
		NextLayer->device_diff_data, &beta, BiasTensorDesc, device_grad_b));


	checkCUDNN(hipdnnConvolutionBackwardFilter(neuralNetwork->cudnnHandle, &alpha, LastLayer->TensorDesc,
		LastLayer->device_data, TensorDesc, NextLayer->device_diff_data, ConvDesc,
		BwdAlgDesc, neuralNetwork->device_workspace, neuralNetwork->WorkspaceSize,
		&beta, FilterDesc, device_grad_w));

	if (!isFirstLayer)
	{
		checkCUDNN(hipdnnConvolutionBackwardData(neuralNetwork->cudnnHandle, &alpha, FilterDesc,
			device_param_w, TensorDesc, NextLayer->device_diff_data, ConvDesc,
			BwdDataAlgDesc, neuralNetwork->device_workspace, neuralNetwork->WorkspaceSize,
			&beta, LastLayer->TensorDesc, device_diff_data));
	}	
}

inline void ConvolutionLayer::UpdateWeights(float learning_rate)
{
	float alpha = -learning_rate;

	checkCudaErrors(hipblasSaxpy(neuralNetwork->cublasHandle, static_cast<int>(ParamW.size()),
		&alpha, device_grad_w, 1, device_param_w, 1));
	checkCudaErrors(hipblasSaxpy(neuralNetwork->cublasHandle, static_cast<int>(ParamB.size()),
		&alpha, device_grad_b, 1, device_param_b, 1));
}

inline void ConvolutionLayer::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * OutputChannels * OutputHeight * OutputWidth));

	// ����
	checkCudaErrors(hipMalloc(&device_param_w, sizeof(float) * ParamW.size()));
	checkCudaErrors(hipMalloc(&device_param_b, sizeof(float) * ParamB.size()));
	// �ݶ�
	checkCudaErrors(hipMalloc(&device_grad_w, sizeof(float) * ParamW.size()));
	checkCudaErrors(hipMalloc(&device_grad_b, sizeof(float) * ParamB.size()));
	// ���򴫲�����
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * InputNumber));

	// ������ʼ�����ݵ�GPU
	checkCudaErrors(hipMemcpyAsync(device_param_w, &ParamW[0], sizeof(float) * ParamW.size(), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyAsync(device_param_b, &ParamB[0], sizeof(float) * ParamB.size(), hipMemcpyHostToDevice));
}

inline void ConvolutionLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_param_w));
	checkCudaErrors(hipFree(device_param_b));
	checkCudaErrors(hipFree(device_grad_w));
	checkCudaErrors(hipFree(device_grad_b));
	checkCudaErrors(hipFree(device_diff_data));
	checkCudaErrors(hipFree(device_param_w));
	checkCudaErrors(hipFree(device_param_b));

}

inline void ConvolutionLayer::CreateDescriptor(int batchsize)
{
	size_t tempsize;
	// ��������
	checkCUDNN(hipdnnCreateTensorDescriptor(&TensorDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&BiasTensorDesc));
	checkCUDNN(hipdnnCreateFilterDescriptor(&FilterDesc));
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&ConvDesc));

	// ��������
	checkCUDNN(hipdnnSetTensor4dDescriptor(BiasTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, OutputChannels, 1, 1));
	checkCUDNN(hipdnnSetFilter4dDescriptor(FilterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, OutputChannels, InputChannels, KernelSize, KernelSize));
	checkCUDNN(hipdnnSetConvolution2dDescriptor(ConvDesc, Padding, Padding, Stride, Stride, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
	checkCUDNN(hipdnnSetTensor4dDescriptor(TensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchsize, OutputChannels, OutputHeight, OutputWidth));

	// ǰ�򴫲��㷨����
	checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(neuralNetwork->cudnnHandle, LastLayer->TensorDesc, FilterDesc, ConvDesc, TensorDesc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &FwdAlgDesc));
	checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(neuralNetwork->cudnnHandle, LastLayer->TensorDesc, FilterDesc, ConvDesc, TensorDesc, FwdAlgDesc, &tempsize));
	neuralNetwork->WorkspaceSize = max(neuralNetwork->WorkspaceSize, tempsize);

	// ���򴫲��㷨����
	checkCUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(neuralNetwork->cudnnHandle, LastLayer->TensorDesc, TensorDesc, ConvDesc, FilterDesc, HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &BwdAlgDesc));
	checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(neuralNetwork->cudnnHandle, LastLayer->TensorDesc, TensorDesc, ConvDesc, FilterDesc, BwdAlgDesc, &tempsize));
	neuralNetwork->WorkspaceSize = max(neuralNetwork->WorkspaceSize, tempsize);
	checkCUDNN(hipdnnGetConvolutionBackwardDataAlgorithm(neuralNetwork->cudnnHandle, FilterDesc, TensorDesc, ConvDesc, LastLayer->TensorDesc, HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &BwdDataAlgDesc));
	checkCUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(neuralNetwork->cudnnHandle, FilterDesc, TensorDesc, ConvDesc, LastLayer->TensorDesc, BwdDataAlgDesc, &tempsize));
	neuralNetwork->WorkspaceSize = max(neuralNetwork->WorkspaceSize, tempsize);
}

inline void ConvolutionLayer::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(TensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(BiasTensorDesc));
	checkCUDNN(hipdnnDestroyFilterDescriptor(FilterDesc));
	checkCUDNN(hipdnnDestroyConvolutionDescriptor(ConvDesc));
}

/*
// MaxPoolLayer
*/
MaxPoolLayer::MaxPoolLayer(NeuralNetwork *neuralnetwork, Layer *lastlayer, int size, int stride)
{
	InputChannels = OutputChannels = lastlayer->OutputChannels;
	InputWidth = lastlayer->OutputWidth;
	InputHeight = lastlayer->OutputHeight;
	OutputWidth = InputWidth / stride;
	OutputHeight = InputHeight / stride;
	InputNumber = InputWidth * InputHeight * InputChannels;
	OutputNumber = OutputWidth * OutputHeight * OutputChannels;
	KernelSize = size;
	Stride = stride;
	Padding = 0;

	neuralNetwork = neuralnetwork;
	LastLayer = lastlayer;
	lastlayer->NextLayer = this;

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);
}

MaxPoolLayer::~MaxPoolLayer()
{
	DestroyDescriptor();
	deviceFree();
}

inline void MaxPoolLayer::ForwardPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;
	checkCUDNN(hipdnnPoolingForward(neuralNetwork->cudnnHandle, PoolDesc, &alpha, LastLayer->TensorDesc,
		LastLayer->device_data, &beta, TensorDesc, device_data));
}

inline void MaxPoolLayer::BackPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;
	if (!isFirstLayer)
	{
		checkCUDNN(hipdnnPoolingBackward(neuralNetwork->cudnnHandle, PoolDesc, &alpha,
			TensorDesc, device_data, TensorDesc, NextLayer->device_diff_data,
			LastLayer->TensorDesc, LastLayer->device_data, &beta, LastLayer->TensorDesc, device_diff_data));
	}
	
}

inline void MaxPoolLayer::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * OutputNumber));
	// ���򴫲�����
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * InputNumber));
}

inline void MaxPoolLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_diff_data));
}

inline void MaxPoolLayer::CreateDescriptor(int batchsize)
{
	// ����������
	checkCUDNN(hipdnnCreateTensorDescriptor(&TensorDesc));
	checkCUDNN(hipdnnCreatePoolingDescriptor(&PoolDesc));

	// ����������
	checkCUDNN(hipdnnSetPooling2dDescriptor(PoolDesc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN, KernelSize, KernelSize, 0, 0, Stride, Stride));
	checkCUDNN(hipdnnSetTensor4dDescriptor(TensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchsize, OutputChannels, OutputHeight, OutputWidth));
}

inline void MaxPoolLayer::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(TensorDesc));
	checkCUDNN(hipdnnDestroyPoolingDescriptor(PoolDesc));
}


/*
// DataSet
*/
DataSet::DataSet()
{
	size_t width, height;
	printf("Reading input data\n");
	
	size_t train_size = ReadUByteDataset(TrainingSetName.c_str(), TrainingLabelsName.c_str(), nullptr, nullptr, width, height);
	size_t test_size = ReadUByteDataset(TestSetName.c_str(), TestLabelsName.c_str(), nullptr, nullptr, width, height);
	TrainSize = train_size;
	TestSize = test_size;
	if (train_size == 0)
		exit(1);

	InputChannels = OutputChannels = 1;
	InputWidth = OutputWidth = width;
	InputHeight = OutputHeight = height;
	InputNumber = InputHeight * InputWidth * InputChannels;
	OutputNumber = OutputHeight * OutputWidth * OutputChannels;
	Padding = 0;
	KernelSize = 1;
	Stride = 1;

	TrainSet.resize(train_size * OutputNumber);
	TrainLabels.resize(train_size);
	TestSet.resize(test_size * OutputNumber);
	TestLabels.resize(test_size);

	if (ReadUByteDataset(TrainingSetName.c_str(), TrainingLabelsName.c_str(), &TrainSet[0], &TrainLabels[0], width, height) != train_size)
		exit(2);
	if (ReadUByteDataset(TestSetName.c_str(), TestLabelsName.c_str(), &TestSet[0], &TestLabels[0], width, height) != test_size)
		exit(3);

	printf("Done. Training dataset size: %d, Test dataset size: %d\n", (int)train_size, (int)test_size);
	
	// Normalize training set to be in [0,1]
	printf("Normalizing training data...\n");
	TrainSet_float.resize(TrainSet.size());
	TrainLabels_float.resize(TrainLabels.size());
	for (size_t i = 0; i < train_size * OutputNumber; ++i)
		TrainSet_float[i] = (float)TrainSet[i] / 255.0f;

	for (size_t i = 0; i < train_size; ++i)
		TrainLabels_float[i] = (float)TrainLabels[i];

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);
}

DataSet::~DataSet()
{
	DestroyDescriptor();
	deviceFree();
}

//inline void DataSet::ForwardPropagate()
//{
//	static int iter = 0;
//	iter++;
//	int imageid = iter % (TrainSize / BATCH_SIZE);
//	/*checkCudaErrors(hipMemcpyAsync(device_data, &((TrainSet_float)[imageid * BATCH_SIZE * OutputNumber]),
//		sizeof(float) * BATCH_SIZE * OutputNumber, hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemcpyAsync(device_labels, &((TrainLabels_float)[imageid * BATCH_SIZE]),
//		sizeof(float) * BATCH_SIZE, hipMemcpyHostToDevice));*/
//	checkCudaErrors(hipMemcpyAsync(device_data, &((TrainSet_float)[imageid * BATCH_SIZE * OutputNumber]),
//		sizeof(float) * BATCH_SIZE * OutputNumber, hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemcpyAsync(device_labels, &((TrainLabels_float)[imageid * BATCH_SIZE]),
//		sizeof(float) * BATCH_SIZE, hipMemcpyHostToDevice));
//}

inline void DataSet::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * OutputNumber));
	checkCudaErrors(hipMalloc(&device_labels, sizeof(float) * batchsize));
}

inline void DataSet::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_labels));
}

inline void DataSet::CreateDescriptor(int batchsize)
{
	// ��������
	checkCUDNN(hipdnnCreateTensorDescriptor(&TensorDesc)); //

	// ��������
	checkCUDNN(hipdnnSetTensor4dDescriptor(TensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchsize, OutputChannels, OutputHeight, OutputWidth));
}

inline void DataSet::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(TensorDesc));
}

/*
// OutputLayer
*/
OutputLayer::OutputLayer(NeuralNetwork *neuralnetwork, Layer *lastlayer)
{
	OutputNumber = InputNumber = lastlayer->OutputNumber;
	OutputHeight = InputHeight = lastlayer->OutputHeight;
	OutputWidth = InputWidth = lastlayer->OutputWidth;
	OutputChannels = InputChannels = lastlayer->OutputChannels;
	Padding = 0;
	KernelSize = 1;
	Stride = 1;

	neuralNetwork = neuralnetwork;
	LastLayer = lastlayer;
	lastlayer->NextLayer = this;

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);
}

OutputLayer::~OutputLayer()
{
	DestroyDescriptor();
	deviceFree();
}

inline void OutputLayer::ForwardPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;
	checkCUDNN(hipdnnSoftmaxForward(neuralNetwork->cudnnHandle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
		&alpha, LastLayer->TensorDesc, LastLayer->device_data, &beta, LastLayer->TensorDesc, device_data));
}

inline void OutputLayer::BackPropagate()
{
	static float scalVal = 1.0f / static_cast<float>(BATCH_SIZE);
	static float alpha = 1.0f, beta = 0.0f;
	// Initialization (using the training error function)
	//checkCudaErrors(hipMemcpyAsync(device_diff_data, device_data, sizeof(float) * BATCH_SIZE * LastLayer->OutputNumber, hipMemcpyDeviceToDevice));

	// Softmax layer
	SoftmaxLossBackprop <<<RoundUp(BATCH_SIZE, BW), BW >>> (neuralNetwork->device_labels, device_data, device_diff_data, LastLayer->OutputNumber, BATCH_SIZE);

	// Accounting for batch size in SGD
	//checkCudaErrors(hipblasSscal(neuralNetwork->cublasHandle, LastLayer->OutputNumber * BATCH_SIZE, &scalVal, device_diff_data, 1));
	checkCudaErrors(hipdnnSoftmaxBackward(neuralNetwork->cudnnHandle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
		&alpha, LastLayer->TensorDesc, device_data, LastLayer->TensorDesc, device_diff_data, &beta, LastLayer->TensorDesc, device_diff_data));// ���һ��device_diff_data ��Ϊdevice_diff_data
}

inline void OutputLayer::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * OutputNumber));
	// ���򴫲�����
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * InputNumber));
	checkCudaErrors(hipMalloc(&device_loss_data, sizeof(float) * batchsize * OutputNumber));
}

inline void OutputLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_diff_data));
	checkCudaErrors(hipFree(device_loss_data));
}

inline void OutputLayer::CreateDescriptor(int batchsize)
{

}

inline void OutputLayer::DestroyDescriptor()
{

}


/*
// NeuralNetwork
*/
NeuralNetwork::NeuralNetwork()
{
	int gpu_num;
	checkCudaErrors(hipGetDeviceCount(&gpu_num));
	if (GPUid < 0 || GPUid >= gpu_num)
	{
		printf("ERROR: Invalid GPU ID %d (There are %d GPUs on this machine)\n", GPUid, gpu_num);
		exit(4);
	}

	checkCudaErrors(hipblasCreate(&cublasHandle));
	checkCUDNN(hipdnnCreate(&cudnnHandle));
}

NeuralNetwork::~NeuralNetwork()
{
	
}

void NeuralNetwork::AddData(DataSet *dataset)
{
	Data = dataset;
}

void NeuralNetwork::AddLayer(Layer *layer, bool isfirstlayer)
{
	if (isfirstlayer)
	{
		layer->isFirstLayer = true;
	}
	Layers.push_back(layer);
}

void NeuralNetwork::Create()
{
	/*Image = new DataSet();
	Conv1 = new ConvolutionLayer(this, Image, 20, 5);
	Pool1 = new MaxPoolLayer(this, Conv1, 2, 2);
	Conv2 = new ConvolutionLayer(this, Pool1, 50, 5);
	Pool2 = new MaxPoolLayer(this, Conv2, 2, 2);
	FC1 = new FullyConnectedLayer(this, Pool2, 500);
	ACTN1 = new ActivationLayer(this, FC1);
	FC2 = new FullyConnectedLayer(this, ACTN1, 10);
	RSLT = new OutputLayer(this, FC2);*/
	
	device_labels = Data->getLabels();
	checkCudaErrors(hipMalloc(&device_ones, sizeof(float)* BATCH_SIZE));
	FillOnes <<<RoundUp(BATCH_SIZE, BW), BW>>> (device_ones, BATCH_SIZE);
	if (WorkspaceSize > 0)
		checkCudaErrors(hipMalloc(&device_workspace, WorkspaceSize));

}

void NeuralNetwork::Destroy()
{
	delete Data;
	while (!Layers.empty())
	{
		delete Layers.back();
		Layers.pop_back();
	}
	//delete Image;
	//delete Conv1;
	//delete Pool1;
	//delete Conv2;
	//delete Pool2;
	//delete FC1;
	//delete ACTN1;
	//delete FC2;
	//delete RSLT;

	checkCudaErrors(hipFree(device_ones));
	if (device_workspace != nullptr)
		checkCudaErrors(hipFree(device_workspace));
	checkCudaErrors(hipDeviceReset());
}

void NeuralNetwork::Train(int iterations)
{
	printf("Training...\n");
	checkCudaErrors(hipDeviceSynchronize());
	auto t1 = std::chrono::high_resolution_clock::now();

	size_t train_size = Data->getTrainSize();
	float *device_data = Data->getData();

	for (int iter = 0; iter < iterations; ++iter)
	{
		int imageid = iter % (train_size / BATCH_SIZE);
		checkCudaErrors(hipMemcpyAsync(device_data, &((Data->TrainSet_float)[imageid * BATCH_SIZE * Data->getOutputNumber()]),
			sizeof(float) * BATCH_SIZE * Data->getOutputNumber(), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpyAsync(device_labels, &((Data->TrainLabels_float)[imageid * BATCH_SIZE]),
			sizeof(float) * BATCH_SIZE, hipMemcpyHostToDevice));


		// Forward propagation
		ForwardPropagate();

		// Backward propagation
		BackPropagate();

		// Compute learning rate
		float learningRate = static_cast<float>(LEARNING_RATE * pow((1.0 + LR_GAMMA * iter), (-LR_POWER)));
		
		// Update weights
		UpdateWeights(learningRate);
	}
	checkCudaErrors(hipDeviceSynchronize());
	auto t2 = std::chrono::high_resolution_clock::now();

	printf("Iteration time: %f ms\n", std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count() / 1000.0f / iterations);

}


void NeuralNetwork::Test()
{
	float classification_error = 1.0f;

	int classifications = (int)(Data->getTestSize() / 10);

	// Test the resulting neural network's classification

	// Initialize a TrainingContext structure for testing (different batch size)
	NeuralNetwork test_nn;

	// Ensure correct workspaceSize is allocated for testing
	if (WorkspaceSize < test_nn.WorkspaceSize)
	{
		checkCudaErrors(hipFree(device_workspace));
		checkCudaErrors(hipMalloc(&device_workspace, test_nn.WorkspaceSize));
	}

	int num_errors = 0;
	for (int i = 0; i < classifications; ++i)
	{
		int output_number = Data->getOutputNumber();
		float *device_data = Data->getData();
		std::vector<float> data(output_number);
		// Normalize image to be in [0,1]
		for (int j = 0; j < output_number; ++j)
			data[j] = (float)Data->TestSet[i * output_number + j] / 255.0f;

		checkCudaErrors(hipMemcpyAsync(device_data, &data[0], sizeof(float) * output_number, hipMemcpyHostToDevice));

		// Forward propagate test image
		Predict();

		// Perform classification
		std::vector<float> class_vec(10);

		// Copy back result
		checkCudaErrors(hipMemcpy(&class_vec[0], Layers.back()->getData(), sizeof(float) * 10, hipMemcpyDeviceToHost));

		// Determine classification according to maximal response
		int chosen = 0;
		for (int id = 1; id < 10; ++id)
		{
			if (class_vec[chosen] < class_vec[id]) chosen = id;
		}

		if (chosen != Data->TestLabels[i])
			++num_errors;
	}
	classification_error = (float)num_errors / (float)classifications;

	printf("Classification result: %.2f%% error (used %d images)\n", classification_error * 100.0f, (int)classifications);
	
}


void NeuralNetwork::ForwardPropagate()
{
	checkCudaErrors(hipSetDevice(GPUid));
	
	int count = Layers.size();
	for (int i = 0; i < count; i++)
	{
		(Layers[i])->ForwardPropagate();
	}
}

void NeuralNetwork::Predict()
{
	checkCudaErrors(hipSetDevice(GPUid));

	int count = Layers.size();
	for (int i = 0; i < count; i++)
	{
		(Layers[i])->Predict();
	}
}

void NeuralNetwork::BackPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;

	for (signed int i = Layers.size() - 1; i >= 0; i--)
	{
		Layers[i]->BackPropagate();
	}
}

void NeuralNetwork::UpdateWeights(float learning_rate)
{
	//float alpha = -learning_rate;

	checkCudaErrors(hipSetDevice(GPUid));

	int count = Layers.size();
	for (int i = 0; i < count; i++)
	{
		Layers[i]->UpdateWeights(learning_rate);
	}

}

ResidualBlock::ResidualBlock(NeuralNetwork *neuralnetwork, Layer *lastlayer)
{
	neuralNetwork = neuralnetwork;

	
	BranchLayer *branchlayer = new BranchLayer(neuralNetwork, lastlayer);
	device_branch_data = branchlayer->device_data;
	branchlayer->ResBlock = this;
	neuralNetwork->AddLayer(branchlayer);

	neuralNetwork->AddLayer(new ConvolutionLayer(neuralNetwork, branchlayer, 50, 3, 1));
	neuralNetwork->AddLayer(new ActivationLayer(neuralNetwork, neuralNetwork->Layers.back()));
	neuralNetwork->AddLayer(new ConvolutionLayer(neuralNetwork, neuralNetwork->Layers.back(), 50, 3, 1));

	LastLayer = neuralNetwork->Layers.back();
	LastLayer->NextLayer = this;


	OutputNumber = InputNumber = LastLayer->OutputNumber;
	OutputChannels = InputChannels = LastLayer->InputChannels;
	OutputWidth = InputWidth = LastLayer->OutputWidth;
	OutputHeight = InputHeight = LastLayer->OutputHeight;
	KernelSize = 0;
	Padding = 0;
	Stride = 1;

	deviceMalloc(BATCH_SIZE);
	CreateDescriptor(BATCH_SIZE);
}

ResidualBlock::~ResidualBlock()
{
	deviceFree();
	DestroyDescriptor();
}

inline void ResidualBlock::ForwardPropagate() 
{
	static float alpha = 1.0;

	checkCudaErrors(hipblasScopy(neuralNetwork->cublasHandle, InputNumber * BATCH_SIZE, 
		LastLayer->device_data, 1, device_data, 1));
	checkCudaErrors(hipblasSaxpy(neuralNetwork->cublasHandle, InputNumber * BATCH_SIZE,
		&alpha, device_branch_data, 1, device_data, 1));

}
inline void ResidualBlock::BackPropagate() 
{
	device_diff_data = NextLayer->device_diff_data;

}
inline void ResidualBlock::UpdateWeights(float learning_rate) 
{
	
}

inline void ResidualBlock::deviceMalloc(int batchsize)
{
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * OutputNumber));
}

inline void ResidualBlock::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
}

inline void ResidualBlock::CreateDescriptor(int batchsize)
{
	TensorDesc = LastLayer->TensorDesc;
}

inline void ResidualBlock::DestroyDescriptor()
{

}





BranchLayer::BranchLayer(NeuralNetwork *neuralnetwork, Layer *lastlayer)
{
	neuralNetwork = neuralnetwork;
	LastLayer = lastlayer;
	LastLayer->NextLayer = this;

	device_data = LastLayer->device_data;

	OutputNumber = InputNumber = lastlayer->OutputNumber;
	OutputChannels = InputChannels = lastlayer->InputChannels;
	OutputWidth = InputWidth = lastlayer->OutputWidth;
	OutputHeight = InputHeight = lastlayer->OutputHeight;
	KernelSize = 0;
	Padding = 0;
	Stride = 1;

	deviceMalloc(BATCH_SIZE);
	CreateDescriptor(BATCH_SIZE);
}

BranchLayer::~BranchLayer()
{
	deviceFree();
	DestroyDescriptor();
}

inline void BranchLayer::ForwardPropagate()
{
	device_data = LastLayer->device_data;
}
inline void BranchLayer::BackPropagate()
{
	static float alpha = 1.0;
	checkCudaErrors(hipblasScopy(neuralNetwork->cublasHandle, InputNumber,
		NextLayer->device_diff_data, 1, device_diff_data, 1));
	if (ResBlock != nullptr)
	{
		checkCudaErrors(hipblasSaxpy(neuralNetwork->cublasHandle, InputNumber,
			&alpha, ResBlock->device_diff_data, 1, device_diff_data, 1));
	}
}
inline void BranchLayer::UpdateWeights(float learning_rate)
{

}

inline void BranchLayer::deviceMalloc(int batchsize)
{
	//device_data = LastLayer->device_data;
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * InputNumber));
}

inline void BranchLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_diff_data));
}

inline void BranchLayer::CreateDescriptor(int batchsize)
{
	TensorDesc = LastLayer->TensorDesc;
}

inline void BranchLayer::DestroyDescriptor()
{

}



bool Layer::FromFile(const char *fileprefix)
{
	std::stringstream ssf, ssbf;
	ssf << fileprefix << ".bin";
	ssbf << fileprefix << ".bias.bin";

	// Read weights file
	FILE *fp = fopen(ssf.str().c_str(), "rb");
	if (!fp)
	{
		printf("ERROR: Cannot open file %s\n", ssf.str().c_str());
		return false;
	}
	fread(&ParamW[0], sizeof(float), ParamW.size(), fp);
	fclose(fp);

	// Read bias file
	fp = fopen(ssbf.str().c_str(), "rb");
	if (!fp)
	{
		printf("ERROR: Cannot open file %s\n", ssbf.str().c_str());
		return false;
	}
	fread(&ParamB[0], sizeof(float), ParamB.size(), fp);
	fclose(fp);
	return true;
}

void Layer::ToFile(const char *fileprefix)
{
	std::stringstream ssf, ssbf;
	ssf << fileprefix << ".bin";
	ssbf << fileprefix << ".bias.bin";

	// Write weights file
	FILE *fp = fopen(ssf.str().c_str(), "wb");
	if (!fp)
	{
		printf("ERROR: Cannot open file %s\n", ssf.str().c_str());
		exit(2);
	}
	fwrite(&ParamW[0], sizeof(float), ParamW.size(), fp);
	fclose(fp);

	// Write bias file
	fp = fopen(ssbf.str().c_str(), "wb");
	if (!fp)
	{
		printf("ERROR: Cannot open file %s\n", ssbf.str().c_str());
		exit(2);
	}
	fwrite(&ParamB[0], sizeof(float), ParamB.size(), fp);
	fclose(fp);
}





BatchNormLayer::BatchNormLayer(NeuralNetwork *neuralnetwork, Layer *lastlayer)
{
	neuralNetwork = neuralnetwork;
	LastLayer = lastlayer;
	LastLayer->NextLayer = this;

	OutputNumber = InputNumber = lastlayer->OutputNumber;
	OutputChannels = InputChannels = lastlayer->InputChannels;
	OutputWidth = InputWidth = lastlayer->OutputWidth;
	OutputHeight = InputHeight = lastlayer->OutputHeight;
	KernelSize = 0;
	Padding = 0;
	Stride = 1;	

	deviceMalloc(BATCH_SIZE);
	CreateDescriptor(BATCH_SIZE);
}

BatchNormLayer::~BatchNormLayer()
{
	deviceFree();
	DestroyDescriptor();
}

inline void BatchNormLayer::ForwardPropagate()
{
	static float alpha = 1.0, beta = 0.0;
	device_data = LastLayer->device_data;

	//checkCudaErrors(hipdnnBatchNormalizationForwardTraining(neuralNetwork->cudnnHandle, HIPDNN_BATCHNORM_SPATIAL_PERSISTENT, 
	//	&alpha, &beta, LastLayer->TensorDesc, LastLayer->device_data, TensorDesc, device_data,
	//	));
}

inline void BatchNormLayer::BackPropagate()
{
	device_diff_data = NextLayer->device_diff_data;
}

inline void BatchNormLayer::Predict()
{

}

inline void BatchNormLayer::UpdateWeights(float learning_rate)
{

}

inline void BatchNormLayer::deviceMalloc(int batchsize)
{
	device_data = LastLayer->device_data;
}

inline void BatchNormLayer::deviceFree()
{

}

inline void BatchNormLayer::CreateDescriptor(int batchsize)
{
	checkCUDNN(hipdnnSetTensor4dDescriptor(TensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchsize, OutputChannels, OutputHeight, OutputWidth));
	checkCUDNN(hipdnnSetTensor4dDescriptor(ParamBTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchsize, OutputChannels, OutputHeight, OutputWidth));
}

inline void BatchNormLayer::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(TensorDesc));
}