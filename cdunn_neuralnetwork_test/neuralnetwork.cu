#include "hip/hip_runtime.h"
#include "neuralnetwork.cuh"

/**
* Computes ceil(x / y) for integral nonnegative values.
*/
static inline unsigned int RoundUp(unsigned int nominator, unsigned int denominator)
{
	return (nominator + denominator - 1) / denominator;
}

/**
* Fills a floating-point array with ones.
*
* @param vec The array to fill.
* @param size The number of elements in the array.
*/
__global__ void FillOnes(float *vec, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;

	vec[idx] = 1.0f;
}
/**
* Computes the backpropagation results of the Softmax loss for each result in a batch.
* Uses the softmax values obtained from forward propagation to compute the difference.
*
* @param label The training batch label values.
* @param num_labels The number of possible labels.
* @param batch_size The size of the trained batch.
* @param diff The resulting gradient.
*/
__global__ void SoftmaxLossBackprop(const float *label, int num_labels, int batch_size, float *diff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= batch_size)
		return;

	const int label_value = static_cast<int>(label[idx]);

	// For each item in the batch, decrease the result of the label's value by 1
	diff[idx * num_labels + label_value] -= 1.0f;
}



/*
// FullyConnectedLayer
*/
FullyConnectedLayer::FullyConnectedLayer(Layer *lastlayer,hipblasHandle_t cublashandle, int input_num, int output_num)
{
	InputNumber = input_num;
	OutputNumber = output_num;
	ParamW.resize(input_num * output_num);
	ParamB.resize(output_num);

	LastLayer = lastlayer;
	cublasHandle = cublashandle;

	random_device rd;
	mt19937 gen(rd());
	float wfc = sqrt(3.0f / (InputNumber * OutputNumber));
	std::uniform_real_distribution<> dfc(-wfc, wfc);
	for (auto&& iter : ParamW)
		iter = static_cast<float>(dfc(gen));
	for (auto&& iter : ParamB)
		iter = static_cast<float>(dfc(gen));

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);
}

FullyConnectedLayer::~FullyConnectedLayer()
{
	DestroyDescriptor();
	deviceFree();
}

inline void FullyConnectedLayer::ForwardPropagate(float *device_ones)
{
	static float alpha = 1.0f, beta = 0.0f;
	// Forward propagate neurons using weights (fc1 = pfc1'*pool2)
	checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
		OutputNumber, BATCH_SIZE, InputNumber, &alpha, device_param_w, InputNumber,
		LastLayer->device_data, InputNumber, &beta, device_data, OutputNumber));
	// Add bias using GEMM's "beta" (fc1 += pfc1bias*1_vec')
	checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		OutputNumber, BATCH_SIZE, 1,
		&alpha,
		device_param_b, OutputNumber,
		device_ones, 1,
		&alpha,
		device_data, OutputNumber));
}

inline void FullyConnectedLayer::BackPropagate(float* diff, float* device_ones, bool isFirstLayer)
{
	static float alpha = 1.0f, beta = 0.0f;
	// Compute derivative with respect to weights: gfc2 = (fc1relu * dfc2smax')
	checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, InputNumber, OutputNumber, BATCH_SIZE,
		&alpha, LastLayer->device_data, InputNumber, diff, OutputNumber, &beta, device_grad_w, InputNumber));
	// Compute derivative with respect to bias: gfc2bias = dfc2smax * 1_vec
	checkCudaErrors(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, OutputNumber, BATCH_SIZE,
		&alpha, diff, OutputNumber, device_ones, 1, &beta, device_grad_b, 1));
	// Compute derivative with respect to data (for previous layer): pfc2*dfc2smax (500x10*10xN)
	if (!isFirstLayer)
	{
		checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, InputNumber, BATCH_SIZE, OutputNumber,
			&alpha, device_param_w, InputNumber, diff, OutputNumber, &beta, device_diff_data, InputNumber));
	}
}

inline void FullyConnectedLayer::UpdateWeights(float learning_rate)
{
	float alpha = -learning_rate;
	checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(ParamW.size()),
		&alpha, device_grad_w, 1, device_param_w, 1));
	checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(ParamB.size()),
		&alpha, device_grad_b, 1, device_param_b, 1));
}

inline void FullyConnectedLayer::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * OutputNumber));	// GPU�и����ݿ��ٿռ�
																							// ����
	checkCudaErrors(hipMalloc(&device_param_w, sizeof(float) * ParamW.size()));			// GPU�и�����w���ٿռ�
	checkCudaErrors(hipMalloc(&device_param_b, sizeof(float) * ParamB.size()));			// GPU�и�����b���ٿռ�
																							// �ݶ�
	checkCudaErrors(hipMalloc(&device_grad_w, sizeof(float) * ParamW.size()));				// GPU�и��ݶ�w���ٿռ�
	checkCudaErrors(hipMalloc(&device_grad_b, sizeof(float) * ParamB.size()));				// GPU�и��ݶ�b���ٿռ�
																							// ���򴫲�����
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * InputNumber));

	// ������ʼ�����ݵ�GPU
	checkCudaErrors(hipMemcpyAsync(device_param_w, &ParamW[0], sizeof(float) * ParamW.size(), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyAsync(device_param_b, &ParamB[0], sizeof(float) * ParamB.size(), hipMemcpyHostToDevice));
}

inline void FullyConnectedLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_param_w));
	checkCudaErrors(hipFree(device_param_b));
	checkCudaErrors(hipFree(device_grad_w));
	checkCudaErrors(hipFree(device_grad_b));
	checkCudaErrors(hipFree(device_diff_data));
	checkCudaErrors(hipFree(device_param_w));
	checkCudaErrors(hipFree(device_param_b));
}

inline void FullyConnectedLayer::CreateDescriptor(int batchsize)
{
	checkCUDNN(hipdnnCreateTensorDescriptor(&TensorDesc));

	//
	checkCUDNN(hipdnnSetTensor4dDescriptor(TensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchsize, OutputNumber, 1, 1));
}

inline void FullyConnectedLayer::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(TensorDesc));
}


/*
// ActivationLayer
*/
ActivationLayer::ActivationLayer(Layer *lastlayer, hipdnnHandle_t cudnnhandle, int num, hipdnnActivationMode_t mode, hipdnnNanPropagation_t nanopt, double coef)
{
	Number = num;
	ActivationMode = mode;
	NanOption = nanopt;
	Coef = coef;

	LastLayer = lastlayer;
	cudnnHandle = cudnnhandle;

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);
}

ActivationLayer::~ActivationLayer()
{
	DestroyDescriptor();
	deviceFree();
}

inline void ActivationLayer::ForwardPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;
	checkCUDNN(hipdnnActivationForward(cudnnHandle, ActivationDesc, &alpha,
		LastLayer->TensorDesc, LastLayer->device_data, &beta, LastLayer->TensorDesc, device_data));
}

inline void ActivationLayer::BackPropagate(float * diff, bool isFirstLayer)
{
	static float alpha = 1.0f, beta = 0.0f;
	if (!isFirstLayer)
	{
		checkCUDNN(hipdnnActivationBackward(cudnnHandle, ActivationDesc, &alpha,
			LastLayer->TensorDesc, device_data, LastLayer->TensorDesc, diff,
			LastLayer->TensorDesc, LastLayer->device_data, &beta, LastLayer->TensorDesc, device_diff_data));
	}
}

inline void ActivationLayer::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * Number));
	// ���򴫲�����
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * Number));
}

inline void ActivationLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_diff_data));
}

inline void ActivationLayer::CreateDescriptor(int batchsize)
{
	// ����������
	checkCUDNN(hipdnnCreateActivationDescriptor(&ActivationDesc));

	// ����������
	checkCUDNN(hipdnnSetActivationDescriptor(ActivationDesc, ActivationMode, NanOption, Coef));
}

inline void ActivationLayer::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyActivationDescriptor(ActivationDesc));
}



/*
// ConvolutionLayer
*/
ConvolutionLayer::ConvolutionLayer(Layer *lastlayer, hipdnnHandle_t cudnnhandle, hipblasHandle_t cublashandle, int in_channels, int out_channels, int kernel_size, int in_width, int in_height, int padding, int stride)
{
	InputChannels = in_channels;
	OutputChannels = out_channels;
	KernelSize = kernel_size;
	InputWidth = in_width;
	InputHeight = in_height;
	Padding = padding;
	Stride = stride;

	OutputHeight = (InputHeight + 2 * Padding - KernelSize) / Stride + 1;
	OutputWidth = (InputWidth + 2 * Padding - KernelSize) / Stride + 1;
	ParamW.resize(in_channels * kernel_size * kernel_size * out_channels);
	ParamB.resize(out_channels);

	LastTensorDesc = lastlayer->TensorDesc;
	LastLayer = lastlayer;
	cublasHandle = cublashandle;
	cudnnHandle = cudnnhandle;

	random_device rd;
	mt19937 gen(rd());
	float wconv = sqrt(3.0f / (KernelSize * KernelSize * InputChannels));
	std::uniform_real_distribution<> dconv(-wconv, wconv);
	for (auto&& iter : ParamW)
		iter = static_cast<float>(dconv(gen));
	for (auto&& iter : ParamB)
		iter = static_cast<float>(dconv(gen));

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);

}

ConvolutionLayer::~ConvolutionLayer()
{
	DestroyDescriptor();
	deviceFree();
}


inline void ConvolutionLayer::ForwardPropagate(void *workspace, size_t workspacesize)
{
	static float alpha = 1.0f, beta = 0.0f;
	checkCUDNN(hipdnnConvolutionForward(
		cudnnHandle, &alpha, LastLayer->TensorDesc,
		LastLayer->device_data, FilterDesc, device_param_w, ConvDesc,
		FwdAlgDesc, workspace, workspacesize, &beta,
		TensorDesc, device_data));

	checkCUDNN(hipdnnAddTensor(cudnnHandle, &alpha, BiasTensorDesc,
		device_param_b, &alpha, TensorDesc, device_data));
}

inline void ConvolutionLayer::BackPropagate(float *diff, void *workspace, size_t workspacesize, bool isFistLayer)
{
	static float alpha = 1.0f, beta = 0.0f;

	checkCUDNN(hipdnnConvolutionBackwardBias(cudnnHandle, &alpha, TensorDesc,
		diff, &beta, BiasTensorDesc, device_grad_b));


	checkCUDNN(hipdnnConvolutionBackwardFilter(cudnnHandle, &alpha, LastLayer->TensorDesc,
		LastLayer->device_data, TensorDesc, diff, ConvDesc,
		BwdAlgDesc, workspace, workspacesize,
		&beta, FilterDesc, device_grad_w));

	if (!isFistLayer)
	{
		checkCUDNN(hipdnnConvolutionBackwardData(cudnnHandle, &alpha, FilterDesc,
			device_param_w, TensorDesc, diff, ConvDesc,
			BwdDataAlgDesc, workspace, workspacesize,
			&beta, LastLayer->TensorDesc, device_diff_data));
	}	
}

inline void ConvolutionLayer::UpdateWeights(float learning_rate)
{
	float alpha = -learning_rate;
	checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(ParamW.size()),
		&alpha, device_grad_w, 1, device_param_w, 1));
	checkCudaErrors(hipblasSaxpy(cublasHandle, static_cast<int>(ParamB.size()),
		&alpha, device_grad_b, 1, device_param_b, 1));
}

inline void ConvolutionLayer::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * OutputChannels * OutputHeight * OutputWidth));

	// ����
	checkCudaErrors(hipMalloc(&device_param_w, sizeof(float) * ParamW.size()));
	checkCudaErrors(hipMalloc(&device_param_b, sizeof(float) * ParamB.size()));
	// �ݶ�
	checkCudaErrors(hipMalloc(&device_grad_w, sizeof(float) * ParamW.size()));
	checkCudaErrors(hipMalloc(&device_grad_b, sizeof(float) * ParamB.size()));
	// ���򴫲�����
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * OutputChannels * InputWidth * InputHeight));

	// ������ʼ�����ݵ�GPU
	checkCudaErrors(hipMemcpyAsync(device_param_w, &ParamW[0], sizeof(float) * ParamW.size(), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyAsync(device_param_b, &ParamB[0], sizeof(float) * ParamB.size(), hipMemcpyHostToDevice));
}

inline void ConvolutionLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_param_w));
	checkCudaErrors(hipFree(device_param_b));
	checkCudaErrors(hipFree(device_grad_w));
	checkCudaErrors(hipFree(device_grad_b));
	checkCudaErrors(hipFree(device_diff_data));
	checkCudaErrors(hipFree(device_param_w));
	checkCudaErrors(hipFree(device_param_b));

}

inline void ConvolutionLayer::CreateDescriptor(int batchsize)
{
	size_t tempsize;
	// ��������
	checkCUDNN(hipdnnCreateTensorDescriptor(&TensorDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&BiasTensorDesc));
	checkCUDNN(hipdnnCreateFilterDescriptor(&FilterDesc));
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&ConvDesc));

	// ��������
	checkCUDNN(hipdnnSetTensor4dDescriptor(BiasTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, OutputChannels, 1, 1));
	checkCUDNN(hipdnnSetFilter4dDescriptor(FilterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, OutputChannels, InputChannels, KernelSize, KernelSize));
	checkCUDNN(hipdnnSetConvolution2dDescriptor(ConvDesc, Padding, Padding, Stride, Stride, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
	checkCUDNN(hipdnnSetTensor4dDescriptor(TensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchsize, OutputChannels, OutputHeight, OutputWidth));

	// ǰ�򴫲��㷨����
	checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnnHandle, LastTensorDesc, FilterDesc, ConvDesc, TensorDesc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &FwdAlgDesc));
	checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle, LastTensorDesc, FilterDesc, ConvDesc, TensorDesc, FwdAlgDesc, &tempsize));
	WorkspaceSize = max(WorkspaceSize, tempsize);

	// ���򴫲��㷨����
	checkCUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(cudnnHandle, LastTensorDesc, TensorDesc, ConvDesc, FilterDesc, HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &BwdAlgDesc));
	checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cudnnHandle, LastTensorDesc, TensorDesc, ConvDesc, FilterDesc, BwdAlgDesc, &tempsize));
	WorkspaceSize = max(WorkspaceSize, tempsize);
	checkCUDNN(hipdnnGetConvolutionBackwardDataAlgorithm(cudnnHandle, FilterDesc, TensorDesc, ConvDesc, LastTensorDesc, HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &BwdDataAlgDesc));
	checkCUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnnHandle, FilterDesc, TensorDesc, ConvDesc, LastTensorDesc, BwdDataAlgDesc, &tempsize));
	WorkspaceSize = max(WorkspaceSize, tempsize);
}

inline void ConvolutionLayer::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(TensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(BiasTensorDesc));
	checkCUDNN(hipdnnDestroyFilterDescriptor(FilterDesc));
	checkCUDNN(hipdnnDestroyConvolutionDescriptor(ConvDesc));
}

/*
// MaxPoolLayer
*/
MaxPoolLayer::MaxPoolLayer(Layer *lastlayer, hipdnnHandle_t cudnnhandle, int size, int stride, ConvolutionLayer &lastConv)
{
	Size = size;
	Stride = stride;
	OutputChannels = lastConv.OutputChannels;
	OutputWidth = (lastConv.OutputWidth / stride);
	OutputHeight = (lastConv.OutputHeight / stride);

	LastLayer = lastlayer;
	cudnnHandle = cudnnhandle;

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);
}

MaxPoolLayer::~MaxPoolLayer()
{
	DestroyDescriptor();
	deviceFree();
}

inline void MaxPoolLayer::ForwardPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;
	checkCUDNN(hipdnnPoolingForward(cudnnHandle, PoolDesc, &alpha, LastLayer->TensorDesc,
		LastLayer->device_data, &beta, TensorDesc, device_data));
}

inline void MaxPoolLayer::BackPropagate(float *diff, bool isFirstLayer)
{
	static float alpha = 1.0f, beta = 0.0f;
	if (!isFirstLayer)
	{
		checkCUDNN(hipdnnPoolingBackward(cudnnHandle, PoolDesc, &alpha,
			TensorDesc, device_data, TensorDesc, diff,
			LastLayer->TensorDesc, LastLayer->device_data, &beta, LastLayer->TensorDesc, device_diff_data));
	}
	
}

inline void MaxPoolLayer::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * OutputChannels * OutputHeight * OutputWidth));
	// ���򴫲�����
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * OutputChannels * OutputHeight * OutputWidth));
}

inline void MaxPoolLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_diff_data));
}

inline void MaxPoolLayer::CreateDescriptor(int batchsize)
{
	// ����������
	checkCUDNN(hipdnnCreateTensorDescriptor(&TensorDesc));
	checkCUDNN(hipdnnCreatePoolingDescriptor(&PoolDesc));

	// ����������
	checkCUDNN(hipdnnSetPooling2dDescriptor(PoolDesc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN, Size, Size, 0, 0, Stride, Stride));
	checkCUDNN(hipdnnSetTensor4dDescriptor(TensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchsize, OutputChannels, OutputHeight, OutputWidth));
}

inline void MaxPoolLayer::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(TensorDesc));
	checkCUDNN(hipdnnDestroyPoolingDescriptor(PoolDesc));
}


/*
// DataSet
*/
DataSet::DataSet()
{
	printf("Reading input data\n");

	size_t train_size = ReadUByteDataset(TrainingSetName.c_str(), TrainingLabelsName.c_str(), nullptr, nullptr, Width, Height);
	size_t test_size = ReadUByteDataset(TestSetName.c_str(), TestLabelsName.c_str(), nullptr, nullptr, Width, Height);
	TrainSize = train_size;
	TestSize = test_size;
	if (train_size == 0)
		exit(1);

	TrainSet.resize(train_size * Width * Height * Channels);
	TrainLabels.resize(train_size);
	TestSet.resize(test_size * Width * Height * Channels);
	TestLabels.resize(test_size);

	if (ReadUByteDataset(TrainingSetName.c_str(), TrainingLabelsName.c_str(), &TrainSet[0], &TrainLabels[0], Width, Height) != train_size)
		exit(2);
	if (ReadUByteDataset(TestSetName.c_str(), TestLabelsName.c_str(), &TestSet[0], &TestLabels[0], Width, Height) != test_size)
		exit(3);

	printf("Done. Training dataset size: %d, Test dataset size: %d\n", (int)train_size, (int)test_size);
	
	// Normalize training set to be in [0,1]
	printf("Normalizing training data...\n");
	TrainSet_float.resize(TrainSet.size());
	TrainLabels_float.resize(TrainLabels.size());
	for (size_t i = 0; i < train_size * Channels * Width * Height; ++i)
		TrainSet_float[i] = (float)TrainSet[i] / 255.0f;

	for (size_t i = 0; i < train_size; ++i)
		TrainLabels_float[i] = (float)TrainLabels[i];

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);
}

DataSet::~DataSet()
{
	DestroyDescriptor();
	deviceFree();
}

inline void DataSet::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * Channels * Height * Width));
	checkCudaErrors(hipMalloc(&device_labels, sizeof(float) * batchsize));
}

inline void DataSet::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_labels));
}

inline void DataSet::CreateDescriptor(int batchsize)
{
	// ��������
	checkCUDNN(hipdnnCreateTensorDescriptor(&TensorDesc)); //

														  // ��������
	checkCUDNN(hipdnnSetTensor4dDescriptor(TensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchsize, Channels, Height, Width));
}

inline void DataSet::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(TensorDesc));
}

/*
// OutputLayer
*/
OutputLayer::OutputLayer(Layer *lastlayer, hipdnnHandle_t cudnnhandle, hipblasHandle_t cublashandle, int num)
{
	Number = num;

	LastLayer = lastlayer;
	cudnnHandle = cudnnhandle;
	cublasHandle = cublashandle;

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);
}

OutputLayer::~OutputLayer()
{
	DestroyDescriptor();
	deviceFree();
}

inline void OutputLayer::ForwardPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;
	checkCUDNN(hipdnnSoftmaxForward(cudnnHandle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
		&alpha, LastLayer->TensorDesc, LastLayer->device_data, &beta, LastLayer->TensorDesc, device_data));
}

inline void OutputLayer::BackPropagate(float* device_labels)
{
	static float scalVal = 1.0f / static_cast<float>(BATCH_SIZE);

	// Initialization (using the training error function)
	checkCudaErrors(hipMemcpyAsync(device_diff_data, device_data, sizeof(float) * BATCH_SIZE * LastLayer->OutputNumber, hipMemcpyDeviceToDevice));

	// Softmax layer
	SoftmaxLossBackprop <<<RoundUp(BATCH_SIZE, BW), BW>>> (device_labels, LastLayer->OutputNumber, BATCH_SIZE, device_diff_data);

	// Accounting for batch size in SGD
	checkCudaErrors(hipblasSscal(cublasHandle, LastLayer->OutputNumber * BATCH_SIZE, &scalVal, device_diff_data, 1));

}

inline void OutputLayer::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * Number));
	// ���򴫲�����
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * Number));
	checkCudaErrors(hipMalloc(&device_loss_data, sizeof(float) * batchsize * Number));
}

inline void OutputLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_diff_data));
	checkCudaErrors(hipFree(device_loss_data));
}

inline void OutputLayer::CreateDescriptor(int batchsize)
{

}

inline void OutputLayer::DestroyDescriptor()
{

}

/*
// NeuralNetwork
*/
NeuralNetwork::NeuralNetwork()
{
	int gpu_num;
	checkCudaErrors(hipGetDeviceCount(&gpu_num));
	if (GPUid < 0 || GPUid >= gpu_num)
	{
		printf("ERROR: Invalid GPU ID %d (There are %d GPUs on this machine)\n", GPUid, gpu_num);
		exit(4);
	}

	checkCudaErrors(hipblasCreate(&cublasHandle));
	checkCUDNN(hipdnnCreate(&cudnnHandle));
}

void NeuralNetwork::Create()
{
	Image = new DataSet();
	Conv1 = new ConvolutionLayer(Image, cudnnHandle, cublasHandle, Image->Channels, 20, 5, Image->Height, Image->Width);
	Pool1 = new MaxPoolLayer(Conv1, cudnnHandle, 2, 2, *Conv1);
	Conv2 = new ConvolutionLayer(Pool1, cudnnHandle, cublasHandle, Conv1->OutputChannels, 50, 5, Conv1->OutputWidth / Pool1->Stride, Conv1->OutputHeight / Pool1->Stride);
	Pool2 = new MaxPoolLayer(Conv2, cudnnHandle, 2, 2, *Conv2);
	FC1 = new FullyConnectedLayer(Pool2, cublasHandle, (Conv2->OutputChannels * Conv2->OutputWidth * Conv2->OutputHeight) / (Pool2->Stride * Pool2->Stride), 500);
	ACTN1 = new ActivationLayer(FC1,cudnnHandle, FC1->OutputNumber);
	FC2 = new FullyConnectedLayer(ACTN1, cublasHandle, FC1->OutputNumber, 10);
	RSLT = new OutputLayer(FC2, cudnnHandle, cublasHandle, FC2->OutputNumber);
	
	checkCudaErrors(hipMalloc(&device_ones, sizeof(float)* BATCH_SIZE));
	FillOnes <<<RoundUp(BATCH_SIZE, BW), BW>>> (device_ones, BATCH_SIZE);
	WorkspaceSize = max(Conv1->WorkspaceSize, Conv2->WorkspaceSize);
	if (WorkspaceSize > 0)
		checkCudaErrors(hipMalloc(&device_workspace, WorkspaceSize));

}

void NeuralNetwork::Destroy()
{
	delete Image;
	delete Conv1;
	delete Pool1;
	delete Conv2;
	delete Pool2;
	delete FC1;
	delete ACTN1;
	delete FC2;
	delete RSLT;

	checkCudaErrors(hipFree(device_ones));
	if (device_workspace != nullptr)
		checkCudaErrors(hipFree(device_workspace));
	checkCudaErrors(hipDeviceReset());
}

void NeuralNetwork::Train(int iterations)
{
	printf("Training...\n");
	checkCudaErrors(hipDeviceSynchronize());
	auto t1 = std::chrono::high_resolution_clock::now();
	size_t train_size = Image->TrainSize;
	for (int iter = 0; iter < iterations; ++iter)
	{
		// Train
		int imageid = iter % (train_size / BATCH_SIZE);

		// Prepare current batch on device
		checkCudaErrors(hipMemcpyAsync(Image->device_data, &((Image->TrainSet_float)[imageid * BATCH_SIZE * Image->Width * Image->Height * Image->Channels]),
			sizeof(float) * BATCH_SIZE * Image->Width *Image->Height * Image->Channels, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpyAsync(Image->device_labels, &((Image->TrainLabels_float)[imageid * BATCH_SIZE]),
			sizeof(float) * BATCH_SIZE, hipMemcpyHostToDevice));

		// Forward propagation
		ForwardPropagate();

		// Backward propagation
		BackPropagate();

		// Compute learning rate
		float learningRate = static_cast<float>(LEARNING_RATE * pow((1.0 + LR_GAMMA * iter), (-LR_POWER)));

		// Update weights
		UpdateWeights(learningRate);
	}
	checkCudaErrors(hipDeviceSynchronize());
	auto t2 = std::chrono::high_resolution_clock::now();

	printf("Iteration time: %f ms\n", std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count() / 1000.0f / iterations);

}


void NeuralNetwork::Test()
{
	float classification_error = 1.0f;

	int classifications = (int)(Image->TestSize);

	// Test the resulting neural network's classification

	// Initialize a TrainingContext structure for testing (different batch size)
	NeuralNetwork test_nn;

	// Ensure correct workspaceSize is allocated for testing
	if (WorkspaceSize < test_nn.WorkspaceSize)
	{
		checkCudaErrors(hipFree(device_workspace));
		checkCudaErrors(hipMalloc(&device_workspace, test_nn.WorkspaceSize));
	}

	int num_errors = 0;
	for (int i = 0; i < classifications; ++i)
	{
		std::vector<float> data(Image->Width * Image->Height);
		// Normalize image to be in [0,1]
		for (int j = 0; j < Image->Width * Image->Height; ++j)
			data[j] = (float)Image->TestSet[i * Image->Width * Image->Height * Image->Channels + j] / 255.0f;

		checkCudaErrors(hipMemcpyAsync(Image->device_data, &data[0], sizeof(float) * Image->Width * Image->Height, hipMemcpyHostToDevice));

		// Forward propagate test image
		ForwardPropagate();

		// Perform classification
		std::vector<float> class_vec(10);

		// Copy back result
		checkCudaErrors(hipMemcpy(&class_vec[0], RSLT->device_data, sizeof(float) * 10, hipMemcpyDeviceToHost));

		// Determine classification according to maximal response
		int chosen = 0;
		for (int id = 1; id < 10; ++id)
		{
			if (class_vec[chosen] < class_vec[id]) chosen = id;
		}

		if (chosen != Image->TestLabels[i])
			++num_errors;
	}
	classification_error = (float)num_errors / (float)classifications;

	printf("Classification result: %.2f%% error (used %d images)\n", classification_error * 100.0f, (int)classifications);
	
}


void NeuralNetwork::ForwardPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;
	checkCudaErrors(hipSetDevice(GPUid));

	// Conv1 layer
	Conv1->ForwardPropagate(device_workspace, WorkspaceSize);

	// Pool1 layer
	Pool1->ForwardPropagate();

	// Conv2 layer
	Conv2->ForwardPropagate(device_workspace, WorkspaceSize);

	// Pool2 layer
	//checkCUDNN(hipdnnPoolingForward(cudnnHandle, Pool2->PoolDesc, &alpha, Conv2->TensorDesc,
	//	Conv2->device_data, &beta, Pool2->TensorDesc, Pool2->device_data));
	Pool2->ForwardPropagate();

	// FC1 layer
	FC1->ForwardPropagate(device_ones);


	// ReLU activation
	ACTN1->ForwardPropagate();


	// FC2 layer
	FC2->ForwardPropagate(device_ones);

	// Softmax loss
	RSLT->ForwardPropagate();
}

void NeuralNetwork::BackPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;

	// Output layer
	RSLT->BackPropagate(Image->device_labels);

	// FC2 layer
	FC2->BackPropagate(RSLT->device_diff_data, device_ones);

	// ReLU activation
	ACTN1->BackPropagate(FC2->device_diff_data);

	// FC1 layer
	FC1->BackPropagate(ACTN1->device_diff_data, device_ones);

	// Pool2 layer
	Pool2->BackPropagate(FC1->device_diff_data);

	// Conv2 layer
	Conv2->BackPropagate(Pool2->device_diff_data, device_workspace, WorkspaceSize);

	// Pool1 layer
	Pool1->BackPropagate(Conv2->device_diff_data);

	// Conv1 layer
	Conv1->BackPropagate(Pool1->device_diff_data, device_workspace, WorkspaceSize, true);

	// No need for convBackwardData because there are no more layers below
}

void NeuralNetwork::UpdateWeights(float learning_rate)
{
	float alpha = -learning_rate;

	checkCudaErrors(hipSetDevice(GPUid));

	// Conv1
	Conv1->UpdateWeights(learning_rate);

	// Conv2
	Conv2->UpdateWeights(learning_rate);

	// Fully connected 1
	FC1->UpdateWeights(learning_rate);

	// Fully connected 2
	FC2->UpdateWeights(learning_rate);
}





