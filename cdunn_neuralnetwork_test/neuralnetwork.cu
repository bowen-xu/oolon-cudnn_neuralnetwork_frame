#include "hip/hip_runtime.h"
#include "neuralnetwork.cuh"

/**
* Computes ceil(x / y) for integral nonnegative values.
*/
static inline unsigned int RoundUp(unsigned int nominator, unsigned int denominator)
{
	return (nominator + denominator - 1) / denominator;
}

/**
* Fills a floating-point array with ones.
*
* @param vec The array to fill.
* @param size The number of elements in the array.
*/
__global__ void FillOnes(float *vec, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;

	vec[idx] = 1.0f;
}
/**
* Computes the backpropagation results of the Softmax loss for each result in a batch.
* Uses the softmax values obtained from forward propagation to compute the difference.
*
* @param label The training batch label values.
* @param num_labels The number of possible labels.
* @param batch_size The size of the trained batch.
* @param diff The resulting gradient.
*/
__global__ void SoftmaxLossBackprop(const float *label, int num_labels, int batch_size, float *diff)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= batch_size)
		return;

	const int label_value = static_cast<int>(label[idx]);

	// For each item in the batch, decrease the result of the label's value by 1
	diff[idx * num_labels + label_value] -= 1.0f;
}



/*
// FullyConnectedLayer
*/
FullyConnectedLayer::FullyConnectedLayer(NeuralNetwork *neuralnetwork, Layer *lastlayer, int num)
{
	InputNumber = lastlayer->OutputNumber;
	OutputNumber = num;
	InputChannels = OutputChannels = 1;
	Padding = 0;
	Stride = 1;
	KernelSize = 1;

	ParamW.resize(InputNumber * OutputNumber);
	ParamB.resize(OutputNumber);

	neuralNetwork = neuralnetwork;
	LastLayer = lastlayer;
	lastlayer->NextLayer = this;

	random_device rd;
	mt19937 gen(rd());
	float wfc = sqrt(3.0f / (InputNumber * OutputNumber));
	std::uniform_real_distribution<> dfc(-wfc, wfc);
	for (auto&& iter : ParamW)
		iter = static_cast<float>(dfc(gen));
	for (auto&& iter : ParamB)
		iter = static_cast<float>(dfc(gen));

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);
}


FullyConnectedLayer::~FullyConnectedLayer()
{
	DestroyDescriptor();
	deviceFree();
}

inline void FullyConnectedLayer::ForwardPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;
	// Forward propagate neurons using weights (fc1 = pfc1'*pool2)
	checkCudaErrors(hipblasSgemm(neuralNetwork->cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
		OutputNumber, BATCH_SIZE, InputNumber, &alpha, device_param_w, InputNumber,
		LastLayer->device_data, InputNumber, &beta, device_data, OutputNumber));
	// Add bias using GEMM's "beta" (fc1 += pfc1bias*1_vec')
	checkCudaErrors(hipblasSgemm(neuralNetwork->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		OutputNumber, BATCH_SIZE, 1,
		&alpha,
		device_param_b, OutputNumber,
		neuralNetwork->device_ones, 1,
		&alpha,
		device_data, OutputNumber));
}

inline void FullyConnectedLayer::BackPropagate(bool isFirstLayer)
{
	static float alpha = 1.0f, beta = 0.0f;
	// Compute derivative with respect to weights: gfc2 = (fc1relu * dfc2smax')
	checkCudaErrors(hipblasSgemm(neuralNetwork->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, InputNumber, OutputNumber, BATCH_SIZE,
		&alpha, LastLayer->device_data, InputNumber, NextLayer->device_diff_data, OutputNumber, &beta, device_grad_w, InputNumber));
	// Compute derivative with respect to bias: gfc2bias = dfc2smax * 1_vec
	checkCudaErrors(hipblasSgemv(neuralNetwork->cublasHandle, HIPBLAS_OP_N, OutputNumber, BATCH_SIZE,
		&alpha, NextLayer->device_diff_data, OutputNumber, neuralNetwork->device_ones, 1, &beta, device_grad_b, 1));
	// Compute derivative with respect to data (for previous layer): pfc2*dfc2smax (500x10*10xN)
	if (!isFirstLayer)
	{
		checkCudaErrors(hipblasSgemm(neuralNetwork->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, InputNumber, BATCH_SIZE, OutputNumber,
			&alpha, device_param_w, InputNumber, NextLayer->device_diff_data, OutputNumber, &beta, device_diff_data, InputNumber));
	}
}

inline void FullyConnectedLayer::UpdateWeights(float learning_rate)
{
	float alpha = -learning_rate;
	checkCudaErrors(hipblasSaxpy(neuralNetwork->cublasHandle, static_cast<int>(ParamW.size()),
		&alpha, device_grad_w, 1, device_param_w, 1));
	checkCudaErrors(hipblasSaxpy(neuralNetwork->cublasHandle, static_cast<int>(ParamB.size()),
		&alpha, device_grad_b, 1, device_param_b, 1));
}

inline void FullyConnectedLayer::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * OutputNumber));	// GPU�и����ݿ��ٿռ�
																							// ����
	checkCudaErrors(hipMalloc(&device_param_w, sizeof(float) * ParamW.size()));			// GPU�и�����w���ٿռ�
	checkCudaErrors(hipMalloc(&device_param_b, sizeof(float) * ParamB.size()));			// GPU�и�����b���ٿռ�
																							// �ݶ�
	checkCudaErrors(hipMalloc(&device_grad_w, sizeof(float) * ParamW.size()));				// GPU�и��ݶ�w���ٿռ�
	checkCudaErrors(hipMalloc(&device_grad_b, sizeof(float) * ParamB.size()));				// GPU�и��ݶ�b���ٿռ�
																							// ���򴫲�����
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * InputNumber));

	// ������ʼ�����ݵ�GPU
	checkCudaErrors(hipMemcpyAsync(device_param_w, &ParamW[0], sizeof(float) * ParamW.size(), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyAsync(device_param_b, &ParamB[0], sizeof(float) * ParamB.size(), hipMemcpyHostToDevice));
}

inline void FullyConnectedLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_param_w));
	checkCudaErrors(hipFree(device_param_b));
	checkCudaErrors(hipFree(device_grad_w));
	checkCudaErrors(hipFree(device_grad_b));
	checkCudaErrors(hipFree(device_diff_data));
	checkCudaErrors(hipFree(device_param_w));
	checkCudaErrors(hipFree(device_param_b));
}

inline void FullyConnectedLayer::CreateDescriptor(int batchsize)
{
	checkCUDNN(hipdnnCreateTensorDescriptor(&TensorDesc));

	//
	checkCUDNN(hipdnnSetTensor4dDescriptor(TensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchsize, OutputNumber, 1, 1));
}

inline void FullyConnectedLayer::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(TensorDesc));
}


/*
// ActivationLayer
*/
ActivationLayer::ActivationLayer(NeuralNetwork *neuralnetwork, Layer *lastlayer, hipdnnActivationMode_t mode, hipdnnNanPropagation_t nanopt, double coef)
{
	InputNumber = OutputNumber = lastlayer->OutputNumber;
	InputHeight = OutputHeight = lastlayer->OutputHeight;
	InputWidth = OutputWidth = lastlayer->OutputWidth;
	InputChannels = OutputChannels = lastlayer->OutputChannels;
	Padding = 0;
	KernelSize = 1;
	Stride = 1;

	ActivationMode = mode;
	NanOption = nanopt;
	Coef = coef;

	neuralNetwork = neuralnetwork;
	LastLayer = lastlayer;
	lastlayer->NextLayer = this;

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);
}

ActivationLayer::~ActivationLayer()
{
	DestroyDescriptor();
	deviceFree();
}

inline void ActivationLayer::ForwardPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;
	checkCUDNN(hipdnnActivationForward(neuralNetwork->cudnnHandle, ActivationDesc, &alpha,
		LastLayer->TensorDesc, LastLayer->device_data, &beta, LastLayer->TensorDesc, device_data));
}

inline void ActivationLayer::BackPropagate(bool isFirstLayer)
{
	static float alpha = 1.0f, beta = 0.0f;
	if (!isFirstLayer)
	{
		checkCUDNN(hipdnnActivationBackward(neuralNetwork->cudnnHandle, ActivationDesc, &alpha,
			LastLayer->TensorDesc, device_data, LastLayer->TensorDesc, NextLayer->device_diff_data,
			LastLayer->TensorDesc, LastLayer->device_data, &beta, LastLayer->TensorDesc, device_diff_data));
	}
}

inline void ActivationLayer::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * OutputNumber));
	// ���򴫲�����
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * InputNumber));
}

inline void ActivationLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_diff_data));
}

inline void ActivationLayer::CreateDescriptor(int batchsize)
{
	// ����������
	checkCUDNN(hipdnnCreateActivationDescriptor(&ActivationDesc));

	// ����������
	checkCUDNN(hipdnnSetActivationDescriptor(ActivationDesc, ActivationMode, NanOption, Coef));
}

inline void ActivationLayer::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyActivationDescriptor(ActivationDesc));
}



/*
// ConvolutionLayer
*/
ConvolutionLayer::ConvolutionLayer(NeuralNetwork *neuralnetwork, Layer *lastlayer, int output_channels, int kernel_size, int padding, int stride)
{
	InputWidth = lastlayer->OutputWidth;
	InputHeight = lastlayer->OutputHeight;
	OutputHeight = (InputHeight + 2 * padding - kernel_size) / stride + 1;
	OutputWidth = (InputWidth + 2 * padding - kernel_size) / stride + 1;
	InputChannels = lastlayer->OutputChannels;
	OutputChannels = output_channels;
	InputNumber = InputHeight * InputWidth * InputChannels;
	OutputNumber = OutputHeight * OutputWidth * OutputChannels;
	KernelSize = kernel_size;
	Padding = padding;
	Stride = stride;
	
	ParamW.resize(InputChannels * KernelSize * KernelSize * OutputChannels);
	ParamB.resize(OutputChannels);

	neuralNetwork = neuralnetwork;
	LastLayer = lastlayer;
	lastlayer->NextLayer = this;

	random_device rd;
	mt19937 gen(rd());
	float wconv = sqrt(3.0f / (KernelSize * KernelSize * InputChannels));
	std::uniform_real_distribution<> dconv(-wconv, wconv);
	for (auto&& iter : ParamW)
		iter = static_cast<float>(dconv(gen));
	for (auto&& iter : ParamB)
		iter = static_cast<float>(dconv(gen));

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);

}

ConvolutionLayer::~ConvolutionLayer()
{
	DestroyDescriptor();
	deviceFree();
}


inline void ConvolutionLayer::ForwardPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;
	checkCUDNN(hipdnnConvolutionForward(
		neuralNetwork->cudnnHandle, &alpha, LastLayer->TensorDesc,
		LastLayer->device_data, FilterDesc, device_param_w, ConvDesc,
		FwdAlgDesc, neuralNetwork->device_workspace, neuralNetwork->WorkspaceSize, &beta,
		TensorDesc, device_data));

	checkCUDNN(hipdnnAddTensor(neuralNetwork->cudnnHandle, &alpha, BiasTensorDesc,
		device_param_b, &alpha, TensorDesc, device_data));
}

inline void ConvolutionLayer::BackPropagate(bool isFistLayer)
{
	static float alpha = 1.0f, beta = 0.0f;

	checkCUDNN(hipdnnConvolutionBackwardBias(neuralNetwork->cudnnHandle, &alpha, TensorDesc,
		NextLayer->device_diff_data, &beta, BiasTensorDesc, device_grad_b));


	checkCUDNN(hipdnnConvolutionBackwardFilter(neuralNetwork->cudnnHandle, &alpha, LastLayer->TensorDesc,
		LastLayer->device_data, TensorDesc, NextLayer->device_diff_data, ConvDesc,
		BwdAlgDesc, neuralNetwork->device_workspace, neuralNetwork->WorkspaceSize,
		&beta, FilterDesc, device_grad_w));

	if (!isFistLayer)
	{
		checkCUDNN(hipdnnConvolutionBackwardData(neuralNetwork->cudnnHandle, &alpha, FilterDesc,
			device_param_w, TensorDesc, NextLayer->device_diff_data, ConvDesc,
			BwdDataAlgDesc, neuralNetwork->device_workspace, neuralNetwork->WorkspaceSize,
			&beta, LastLayer->TensorDesc, device_diff_data));
	}	
}

inline void ConvolutionLayer::UpdateWeights(float learning_rate)
{
	float alpha = -learning_rate;
	checkCudaErrors(hipblasSaxpy(neuralNetwork->cublasHandle, static_cast<int>(ParamW.size()),
		&alpha, device_grad_w, 1, device_param_w, 1));
	checkCudaErrors(hipblasSaxpy(neuralNetwork->cublasHandle, static_cast<int>(ParamB.size()),
		&alpha, device_grad_b, 1, device_param_b, 1));
}

inline void ConvolutionLayer::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * OutputChannels * OutputHeight * OutputWidth));

	// ����
	checkCudaErrors(hipMalloc(&device_param_w, sizeof(float) * ParamW.size()));
	checkCudaErrors(hipMalloc(&device_param_b, sizeof(float) * ParamB.size()));
	// �ݶ�
	checkCudaErrors(hipMalloc(&device_grad_w, sizeof(float) * ParamW.size()));
	checkCudaErrors(hipMalloc(&device_grad_b, sizeof(float) * ParamB.size()));
	// ���򴫲�����
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * OutputChannels * InputWidth * InputHeight));

	// ������ʼ�����ݵ�GPU
	checkCudaErrors(hipMemcpyAsync(device_param_w, &ParamW[0], sizeof(float) * ParamW.size(), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyAsync(device_param_b, &ParamB[0], sizeof(float) * ParamB.size(), hipMemcpyHostToDevice));
}

inline void ConvolutionLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_param_w));
	checkCudaErrors(hipFree(device_param_b));
	checkCudaErrors(hipFree(device_grad_w));
	checkCudaErrors(hipFree(device_grad_b));
	checkCudaErrors(hipFree(device_diff_data));
	checkCudaErrors(hipFree(device_param_w));
	checkCudaErrors(hipFree(device_param_b));

}

inline void ConvolutionLayer::CreateDescriptor(int batchsize)
{
	size_t tempsize;
	// ��������
	checkCUDNN(hipdnnCreateTensorDescriptor(&TensorDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&BiasTensorDesc));
	checkCUDNN(hipdnnCreateFilterDescriptor(&FilterDesc));
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&ConvDesc));

	// ��������
	checkCUDNN(hipdnnSetTensor4dDescriptor(BiasTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, OutputChannels, 1, 1));
	checkCUDNN(hipdnnSetFilter4dDescriptor(FilterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, OutputChannels, InputChannels, KernelSize, KernelSize));
	checkCUDNN(hipdnnSetConvolution2dDescriptor(ConvDesc, Padding, Padding, Stride, Stride, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
	checkCUDNN(hipdnnSetTensor4dDescriptor(TensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchsize, OutputChannels, OutputHeight, OutputWidth));

	// ǰ�򴫲��㷨����
	checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(neuralNetwork->cudnnHandle, LastLayer->TensorDesc, FilterDesc, ConvDesc, TensorDesc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &FwdAlgDesc));
	checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(neuralNetwork->cudnnHandle, LastLayer->TensorDesc, FilterDesc, ConvDesc, TensorDesc, FwdAlgDesc, &tempsize));
	neuralNetwork->WorkspaceSize = max(neuralNetwork->WorkspaceSize, tempsize);

	// ���򴫲��㷨����
	checkCUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(neuralNetwork->cudnnHandle, LastLayer->TensorDesc, TensorDesc, ConvDesc, FilterDesc, HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &BwdAlgDesc));
	checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(neuralNetwork->cudnnHandle, LastLayer->TensorDesc, TensorDesc, ConvDesc, FilterDesc, BwdAlgDesc, &tempsize));
	neuralNetwork->WorkspaceSize = max(neuralNetwork->WorkspaceSize, tempsize);
	checkCUDNN(hipdnnGetConvolutionBackwardDataAlgorithm(neuralNetwork->cudnnHandle, FilterDesc, TensorDesc, ConvDesc, LastLayer->TensorDesc, HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &BwdDataAlgDesc));
	checkCUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(neuralNetwork->cudnnHandle, FilterDesc, TensorDesc, ConvDesc, LastLayer->TensorDesc, BwdDataAlgDesc, &tempsize));
	neuralNetwork->WorkspaceSize = max(neuralNetwork->WorkspaceSize, tempsize);
}

inline void ConvolutionLayer::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(TensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(BiasTensorDesc));
	checkCUDNN(hipdnnDestroyFilterDescriptor(FilterDesc));
	checkCUDNN(hipdnnDestroyConvolutionDescriptor(ConvDesc));
}

/*
// MaxPoolLayer
*/
MaxPoolLayer::MaxPoolLayer(NeuralNetwork *neuralnetwork, Layer *lastlayer, int size, int stride)
{
	InputChannels = OutputChannels = lastlayer->OutputChannels;
	InputWidth = lastlayer->OutputWidth;
	InputHeight = lastlayer->OutputHeight;
	OutputWidth = InputWidth / stride;
	OutputHeight = InputHeight / stride;
	InputNumber = InputWidth * InputHeight * InputChannels;
	OutputNumber = OutputWidth * OutputHeight * OutputChannels;
	KernelSize = size;
	Stride = stride;
	Padding = 0;

	neuralNetwork = neuralnetwork;
	LastLayer = lastlayer;
	lastlayer->NextLayer = this;

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);
}

MaxPoolLayer::~MaxPoolLayer()
{
	DestroyDescriptor();
	deviceFree();
}

inline void MaxPoolLayer::ForwardPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;
	checkCUDNN(hipdnnPoolingForward(neuralNetwork->cudnnHandle, PoolDesc, &alpha, LastLayer->TensorDesc,
		LastLayer->device_data, &beta, TensorDesc, device_data));
}

inline void MaxPoolLayer::BackPropagate(bool isFirstLayer)
{
	static float alpha = 1.0f, beta = 0.0f;
	if (!isFirstLayer)
	{
		checkCUDNN(hipdnnPoolingBackward(neuralNetwork->cudnnHandle, PoolDesc, &alpha,
			TensorDesc, device_data, TensorDesc, NextLayer->device_diff_data,
			LastLayer->TensorDesc, LastLayer->device_data, &beta, LastLayer->TensorDesc, device_diff_data));
	}
	
}

inline void MaxPoolLayer::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * OutputChannels * OutputHeight * OutputWidth));
	// ���򴫲�����
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * OutputChannels * OutputHeight * OutputWidth));
}

inline void MaxPoolLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_diff_data));
}

inline void MaxPoolLayer::CreateDescriptor(int batchsize)
{
	// ����������
	checkCUDNN(hipdnnCreateTensorDescriptor(&TensorDesc));
	checkCUDNN(hipdnnCreatePoolingDescriptor(&PoolDesc));

	// ����������
	checkCUDNN(hipdnnSetPooling2dDescriptor(PoolDesc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN, KernelSize, KernelSize, 0, 0, Stride, Stride));
	checkCUDNN(hipdnnSetTensor4dDescriptor(TensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchsize, OutputChannels, OutputHeight, OutputWidth));
}

inline void MaxPoolLayer::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(TensorDesc));
	checkCUDNN(hipdnnDestroyPoolingDescriptor(PoolDesc));
}


/*
// DataSet
*/
DataSet::DataSet()
{
	size_t width, height;
	printf("Reading input data\n");
	
	size_t train_size = ReadUByteDataset(TrainingSetName.c_str(), TrainingLabelsName.c_str(), nullptr, nullptr, width, height);
	size_t test_size = ReadUByteDataset(TestSetName.c_str(), TestLabelsName.c_str(), nullptr, nullptr, width, height);
	TrainSize = train_size;
	TestSize = test_size;
	if (train_size == 0)
		exit(1);

	InputChannels = OutputChannels = 1;
	InputWidth = OutputWidth = width;
	InputHeight = OutputHeight = height;
	InputNumber = InputHeight * InputWidth * InputChannels;
	OutputNumber = OutputHeight * OutputWidth * OutputChannels;
	Padding = 0;
	KernelSize = 1;
	Stride = 1;

	TrainSet.resize(train_size * OutputNumber);
	TrainLabels.resize(train_size);
	TestSet.resize(test_size * OutputNumber);
	TestLabels.resize(test_size);

	if (ReadUByteDataset(TrainingSetName.c_str(), TrainingLabelsName.c_str(), &TrainSet[0], &TrainLabels[0], width, height) != train_size)
		exit(2);
	if (ReadUByteDataset(TestSetName.c_str(), TestLabelsName.c_str(), &TestSet[0], &TestLabels[0], width, height) != test_size)
		exit(3);

	printf("Done. Training dataset size: %d, Test dataset size: %d\n", (int)train_size, (int)test_size);
	
	// Normalize training set to be in [0,1]
	printf("Normalizing training data...\n");
	TrainSet_float.resize(TrainSet.size());
	TrainLabels_float.resize(TrainLabels.size());
	for (size_t i = 0; i < train_size * OutputNumber; ++i)
		TrainSet_float[i] = (float)TrainSet[i] / 255.0f;

	for (size_t i = 0; i < train_size; ++i)
		TrainLabels_float[i] = (float)TrainLabels[i];

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);
}

DataSet::~DataSet()
{
	DestroyDescriptor();
	deviceFree();
}

//inline void DataSet::ForwardPropagate()
//{
//	static int iter = 0;
//	iter++;
//	int imageid = iter % (TrainSize / BATCH_SIZE);
//	/*checkCudaErrors(hipMemcpyAsync(device_data, &((TrainSet_float)[imageid * BATCH_SIZE * OutputNumber]),
//		sizeof(float) * BATCH_SIZE * OutputNumber, hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemcpyAsync(device_labels, &((TrainLabels_float)[imageid * BATCH_SIZE]),
//		sizeof(float) * BATCH_SIZE, hipMemcpyHostToDevice));*/
//	checkCudaErrors(hipMemcpyAsync(device_data, &((TrainSet_float)[imageid * BATCH_SIZE * OutputNumber]),
//		sizeof(float) * BATCH_SIZE * OutputNumber, hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemcpyAsync(device_labels, &((TrainLabels_float)[imageid * BATCH_SIZE]),
//		sizeof(float) * BATCH_SIZE, hipMemcpyHostToDevice));
//}

inline void DataSet::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * OutputNumber));
	checkCudaErrors(hipMalloc(&device_labels, sizeof(float) * batchsize));
}

inline void DataSet::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_labels));
}

inline void DataSet::CreateDescriptor(int batchsize)
{
	// ��������
	checkCUDNN(hipdnnCreateTensorDescriptor(&TensorDesc)); //

	// ��������
	checkCUDNN(hipdnnSetTensor4dDescriptor(TensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchsize, OutputChannels, OutputHeight, OutputWidth));
}

inline void DataSet::DestroyDescriptor()
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(TensorDesc));
}

/*
// OutputLayer
*/
OutputLayer::OutputLayer(NeuralNetwork *neuralnetwork, Layer *lastlayer)
{
	OutputNumber = InputNumber = lastlayer->OutputNumber;
	OutputHeight = InputHeight = lastlayer->OutputHeight;
	OutputWidth = InputWidth = lastlayer->OutputWidth;
	OutputChannels = InputChannels = lastlayer->OutputChannels;
	Padding = 0;
	KernelSize = 1;
	Stride = 1;

	neuralNetwork = neuralnetwork;
	LastLayer = lastlayer;
	lastlayer->NextLayer = this;

	CreateDescriptor(BATCH_SIZE);
	deviceMalloc(BATCH_SIZE);
}

OutputLayer::~OutputLayer()
{
	DestroyDescriptor();
	deviceFree();
}

inline void OutputLayer::ForwardPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;
	checkCUDNN(hipdnnSoftmaxForward(neuralNetwork->cudnnHandle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
		&alpha, LastLayer->TensorDesc, LastLayer->device_data, &beta, LastLayer->TensorDesc, device_data));
}

inline void OutputLayer::BackPropagate()
{
	static float scalVal = 1.0f / static_cast<float>(BATCH_SIZE);

	// Initialization (using the training error function)
	checkCudaErrors(hipMemcpyAsync(device_diff_data, device_data, sizeof(float) * BATCH_SIZE * LastLayer->OutputNumber, hipMemcpyDeviceToDevice));

	// Softmax layer
	SoftmaxLossBackprop <<<RoundUp(BATCH_SIZE, BW), BW>>> (neuralNetwork->device_labels, LastLayer->OutputNumber, BATCH_SIZE, device_diff_data);

	// Accounting for batch size in SGD
	checkCudaErrors(hipblasSscal(neuralNetwork->cublasHandle, LastLayer->OutputNumber * BATCH_SIZE, &scalVal, device_diff_data, 1));

}

inline void OutputLayer::deviceMalloc(int batchsize)
{
	// ǰ�򴫲�����
	checkCudaErrors(hipMalloc(&device_data, sizeof(float) * batchsize * OutputNumber));
	// ���򴫲�����
	checkCudaErrors(hipMalloc(&device_diff_data, sizeof(float) * batchsize * OutputNumber));
	checkCudaErrors(hipMalloc(&device_loss_data, sizeof(float) * batchsize * OutputNumber));
}

inline void OutputLayer::deviceFree()
{
	checkCudaErrors(hipFree(device_data));
	checkCudaErrors(hipFree(device_diff_data));
	checkCudaErrors(hipFree(device_loss_data));
}

inline void OutputLayer::CreateDescriptor(int batchsize)
{

}

inline void OutputLayer::DestroyDescriptor()
{

}

/*
// NeuralNetwork
*/
NeuralNetwork::NeuralNetwork()
{
	int gpu_num;
	checkCudaErrors(hipGetDeviceCount(&gpu_num));
	if (GPUid < 0 || GPUid >= gpu_num)
	{
		printf("ERROR: Invalid GPU ID %d (There are %d GPUs on this machine)\n", GPUid, gpu_num);
		exit(4);
	}

	checkCudaErrors(hipblasCreate(&cublasHandle));
	checkCUDNN(hipdnnCreate(&cudnnHandle));
}

void NeuralNetwork::Create()
{
	Image = new DataSet();
	Conv1 = new ConvolutionLayer(this, Image, 20, 5);
	Pool1 = new MaxPoolLayer(this, Conv1, 2, 2);
	Conv2 = new ConvolutionLayer(this, Pool1, 50, 5);
	Pool2 = new MaxPoolLayer(this, Conv2, 2, 2);
	FC1 = new FullyConnectedLayer(this, Pool2, 500);
	ACTN1 = new ActivationLayer(this, FC1);
	FC2 = new FullyConnectedLayer(this, ACTN1, 10);
	RSLT = new OutputLayer(this, FC2);
	
	device_labels = Image->getLabels();
	checkCudaErrors(hipMalloc(&device_ones, sizeof(float)* BATCH_SIZE));
	FillOnes <<<RoundUp(BATCH_SIZE, BW), BW>>> (device_ones, BATCH_SIZE);
	if (WorkspaceSize > 0)
		checkCudaErrors(hipMalloc(&device_workspace, WorkspaceSize));

}

void NeuralNetwork::Destroy()
{
	delete Image;
	delete Conv1;
	delete Pool1;
	delete Conv2;
	delete Pool2;
	delete FC1;
	delete ACTN1;
	delete FC2;
	delete RSLT;

	checkCudaErrors(hipFree(device_ones));
	if (device_workspace != nullptr)
		checkCudaErrors(hipFree(device_workspace));
	checkCudaErrors(hipDeviceReset());
}

void NeuralNetwork::Train(int iterations)
{
	printf("Training...\n");
	checkCudaErrors(hipDeviceSynchronize());
	auto t1 = std::chrono::high_resolution_clock::now();

	size_t train_size = Image->getTrainSize();
	float *device_data = Image->getData();

	for (int iter = 0; iter < iterations; ++iter)
	{
		int imageid = iter % (train_size / BATCH_SIZE);
		checkCudaErrors(hipMemcpyAsync(device_data, &((Image->TrainSet_float)[imageid * BATCH_SIZE * Image->getOutputNumber()]),
			sizeof(float) * BATCH_SIZE * Image->getOutputNumber(), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpyAsync(device_labels, &((Image->TrainLabels_float)[imageid * BATCH_SIZE]),
			sizeof(float) * BATCH_SIZE, hipMemcpyHostToDevice));


		// Forward propagation
		ForwardPropagate();

		// Backward propagation
		BackPropagate();

		// Compute learning rate
		float learningRate = static_cast<float>(LEARNING_RATE * pow((1.0 + LR_GAMMA * iter), (-LR_POWER)));

		// Update weights
		UpdateWeights(learningRate);
	}
	checkCudaErrors(hipDeviceSynchronize());
	auto t2 = std::chrono::high_resolution_clock::now();

	printf("Iteration time: %f ms\n", std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count() / 1000.0f / iterations);

}


void NeuralNetwork::Test()
{
	float classification_error = 1.0f;

	int classifications = (int)(Image->getTestSize());

	// Test the resulting neural network's classification

	// Initialize a TrainingContext structure for testing (different batch size)
	NeuralNetwork test_nn;

	// Ensure correct workspaceSize is allocated for testing
	if (WorkspaceSize < test_nn.WorkspaceSize)
	{
		checkCudaErrors(hipFree(device_workspace));
		checkCudaErrors(hipMalloc(&device_workspace, test_nn.WorkspaceSize));
	}

	int num_errors = 0;
	for (int i = 0; i < classifications; ++i)
	{
		int output_number = Image->getOutputNumber();
		float *device_data = Image->getData();
		std::vector<float> data(output_number);
		// Normalize image to be in [0,1]
		for (int j = 0; j < output_number; ++j)
			data[j] = (float)Image->TestSet[i * output_number + j] / 255.0f;

		checkCudaErrors(hipMemcpyAsync(device_data, &data[0], sizeof(float) * output_number, hipMemcpyHostToDevice));

		// Forward propagate test image
		ForwardPropagate();

		// Perform classification
		std::vector<float> class_vec(10);

		// Copy back result
		checkCudaErrors(hipMemcpy(&class_vec[0], RSLT->getData(), sizeof(float) * 10, hipMemcpyDeviceToHost));

		// Determine classification according to maximal response
		int chosen = 0;
		for (int id = 1; id < 10; ++id)
		{
			if (class_vec[chosen] < class_vec[id]) chosen = id;
		}

		if (chosen != Image->TestLabels[i])
			++num_errors;
	}
	classification_error = (float)num_errors / (float)classifications;

	printf("Classification result: %.2f%% error (used %d images)\n", classification_error * 100.0f, (int)classifications);
	
}


void NeuralNetwork::ForwardPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;
	checkCudaErrors(hipSetDevice(GPUid));
	
	// Conv1 layer
	Conv1->ForwardPropagate();

	// Pool1 layer
	Pool1->ForwardPropagate();

	// Conv2 layer
	Conv2->ForwardPropagate();

	// Pool2 layer
	Pool2->ForwardPropagate();

	// FC1 layer
	FC1->ForwardPropagate();


	// ReLU activation
	ACTN1->ForwardPropagate();


	// FC2 layer
	FC2->ForwardPropagate();


	// Softmax loss
	RSLT->ForwardPropagate();
}

void NeuralNetwork::BackPropagate()
{
	static float alpha = 1.0f, beta = 0.0f;

	// Output layer
	RSLT->BackPropagate();

	// FC2 layer
	FC2->BackPropagate();

	// ReLU activation
	ACTN1->BackPropagate();

	// FC1 layer
	FC1->BackPropagate();

	// Pool2 layer
	Pool2->BackPropagate();

	// Conv2 layer
	Conv2->BackPropagate();

	// Pool1 layer
	Pool1->BackPropagate();

	// Conv1 layer
	Conv1->BackPropagate(true);

	// No need for convBackwardData because there are no more layers below
}

void NeuralNetwork::UpdateWeights(float learning_rate)
{
	float alpha = -learning_rate;

	checkCudaErrors(hipSetDevice(GPUid));

	// Conv1
	Conv1->UpdateWeights(learning_rate);

	// Conv2
	Conv2->UpdateWeights(learning_rate);

	// Fully connected 1
	FC1->UpdateWeights(learning_rate);

	// Fully connected 2
	FC2->UpdateWeights(learning_rate);

}





